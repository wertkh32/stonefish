#pragma once
#include <hip/hip_runtime.h>
#include <stdio.h>

// conjugate gradient solver. by SKH.

#define GPU_SOLVER_MAX_ITER 1000
#define GPU_SOLVER_EPS 0.001
#define BLOCK_SIZE 256

__device__ float dot(float* a, float *b, int n)
{
	float ret = 0;
	for(int i=0;i<n;i++)
	{
		ret += a[i] * b[i];
	}
	return ret;
}

__global__ void gpuCGSolve(float* A, float* x, float* b,
							float* d, float* r, float* q,
							int n)
{

	int id = blockIdx.x *blockDim.x + threadIdx.x;
	
	if(id < n)
	{
		int i = 0;
		float alpha, beta, deltaOld, delta0, deltaNew;

		r[id] = b[id] - dot(&A[id * n], x, n);
		d[id] = r[id];
	
		__syncthreads();

		deltaNew = dot(r,r,n);
		delta0 = deltaNew;

		while(i<GPU_SOLVER_MAX_ITER && deltaNew > GPU_SOLVER_EPS * GPU_SOLVER_EPS * delta0)
		{
			q[id] = dot(&A[id * n],d,n);

			__syncthreads();
		
			alpha = (deltaNew)/dot(d,q,n);
			x[id] += alpha * d[id];

			__syncthreads();

			if(i%50)
			{
				r[id] = b[id] - dot(&A[id * n], x,n);
			}
			else
			{
				r[id] -= alpha * q[id];
			}

			deltaOld = deltaNew;
			deltaNew = dot(r,r,n);
			beta = deltaNew/deltaOld;

			d[id] = r[id] + beta * d[id];

			i = i+1;

			__syncthreads();

		}
	}
}

void CGSolverGPU(float* A, float* x, float* b, int n)
{
	float *gpu_A, *gpu_x, *gpu_b,
			*gpu_d, *gpu_r, *gpu_q;

	int ARR_SIZE = sizeof(float) * n;

	//performance testing/////////////////////////
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	//end///////////////////////////////////////////

	hipMalloc(&gpu_A, sizeof(float) * n * n);

	hipMemcpy(gpu_A, A, sizeof(float) * n * n, hipMemcpyHostToDevice);


	hipMalloc(&gpu_x, ARR_SIZE);
	hipMalloc(&gpu_b, ARR_SIZE);
	hipMemcpy(gpu_x, x, ARR_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(gpu_b, b, ARR_SIZE, hipMemcpyHostToDevice);

	hipMalloc(&gpu_d, ARR_SIZE);
	hipMalloc(&gpu_r, ARR_SIZE);
	hipMalloc(&gpu_q, ARR_SIZE);

	int no_blocks = (n + BLOCK_SIZE - 1)/BLOCK_SIZE;

	gpuCGSolve <<<no_blocks, BLOCK_SIZE>>> (gpu_A, gpu_x, gpu_b, gpu_d, gpu_r, gpu_q, n);

	hipMemcpy(x, gpu_x, ARR_SIZE, hipMemcpyDeviceToHost);

	hipFree(gpu_A);
	hipFree(gpu_x);
	hipFree(gpu_b);
	hipFree(gpu_d);
	hipFree(gpu_r);

	//performance testing///////////////////////
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float timing;
	hipEventElapsedTime( &timing, start, stop );

	printf("Time taken %.4f ms\n",timing);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	//end//////////////////////////////////////////
}
