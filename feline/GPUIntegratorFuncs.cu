#include "hip/hip_runtime.h"
#pragma once
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "GPUDataStructs.cuh"

#define BLOCK_SIZE 256

GPUElement* gpuptrElements;
mat12d* gpuptrWarpK;
float*   gpuptr_x0;//const
float*   gpuptr_xt;//dynamic
float*   gpuptr_vt;//dynamic
float*	 gpuptr_extforces;//dynamic

void
gpuInitVars(int numele, int numnodes)
{
	hipMalloc(&gpuptrElements, numele * sizeof(GPUElement));
	hipMalloc(&gpuptrWarpK, numele * sizeof(mat12d));
	hipMalloc(&gpuptr_x0, numnodes * 3 * sizeof(float));
	hipMalloc(&gpuptr_xt, numnodes * 3 * sizeof(float));
	hipMalloc(&gpuptr_vt, numnodes * 3 * sizeof(float));
	hipMalloc(&gpuptr_extforces, numnodes * 3 * sizeof(float));
}

void
gpuUploadExtForces(float* extforces, int numnodes)
{
	hipMemcpy(gpuptr_extforces, extforces, numnodes*3*sizeof(float),hipMemcpyHostToDevice);
}

__device__
void zeroOut(float* f, int n)
{
	for(int i=0;i<n;i++)
	{
		f[i]=0;
	}
}

__global__
void
timestep(GPUElement* elements, float* x0, float* xt, float*vt,
		float* extforces, int numeles, int numnodes)
{
	
}

