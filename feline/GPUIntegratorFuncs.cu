#include "hip/hip_runtime.h"
#pragma once
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "defines.h"

#ifdef _LINEAR_TET_

#include "GPUDataStructs.cuh"
#include "GPUPolarDecompose.cu"

//#define BLOCK_SIZE 512

#define ALPHA 0.3
#define BETA 0.1

#define MAX_ITER 20
#define EPSIL 0.01

__constant__ float COEFFK, COEFFM, dt;

GPUElement* gpuptrElements;
GPUNode*   gpuptrNodes;
mulData*	gpuptrMulData;
float*   gpuptr_xt;//dynamic
float*   gpuptr_vt;//dynamic
float*	 gpuptr_extforces;//dynamic
float*	 gpuptr_mass;//static
float*	 gpuptr_b;//dynamic
char*	 gpuptr_allowed;

//for CG
float* gpuptrR;
float* gpuptrD;
float* gpuptrQ;
CGVars* gpuptrVars;

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
		system("pause");
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__host__
void
gpuInitVars(int numele, int numnodes)
{
	int numblocksperele = (numele / BLOCK_SIZE) + 1;
	int numblockpernode = (numnodes / NODE_BLOCK_SIZE) + 1;

	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	HANDLE_ERROR( hipMalloc(&gpuptrElements, numblocksperele * sizeof(GPUElement)) );
	HANDLE_ERROR( hipMalloc(&gpuptrMulData, numblocksperele * sizeof(mulData)) );
	HANDLE_ERROR( hipMalloc(&gpuptrNodes, numblockpernode * sizeof(GPUNode)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_xt, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_vt, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_extforces, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_mass, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_b, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_allowed, numnodes * sizeof(char)) );


	HANDLE_ERROR( hipMalloc(&gpuptrR, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptrD, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptrQ, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptrVars, sizeof(CGVars)) );

	float ddt = 1.0/FPS;
	float coeffK = ddt * BETA + ddt * ddt, coeffM = 1 + ddt * ALPHA;
	

	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL("COEFFK"), &coeffK, sizeof(float)) );
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL("COEFFM"), &coeffM, sizeof(float)) );
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL("dt"), &ddt, sizeof(float)) );

		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
			system("pause");
		}
}

__host__
void
gpuUploadVars(GPUElement* gpuElements, GPUNode* gpuNodes,float* xt, 
			  float* vt, float* extforces, float* mass, char* allowed, int numnodes, int numelements)
{
	int numblocksperele = (numelements / BLOCK_SIZE) + 1;
	int numblockpernode = (numnodes / NODE_BLOCK_SIZE) + 1;

	HANDLE_ERROR( hipMemcpy(gpuptrElements, gpuElements, numblocksperele * sizeof(GPUElement), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptrNodes, gpuNodes, numblockpernode * sizeof(GPUNode), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_xt, xt, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_vt, vt, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_extforces, extforces, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_mass, mass, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_allowed, allowed, numnodes * sizeof(char), hipMemcpyHostToDevice) );

		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
			system("pause");
		}
}

__host__
void
gpuDownloadVars(float* xt, int numnodes)
{
	hipMemcpy(xt, gpuptr_xt, numnodes * 3 * sizeof(float), hipMemcpyDeviceToHost);
}

__host__
void
gpuUploadExtForces(float* extforces, int numnodes)
{
	hipMemcpy(gpuptr_extforces, extforces, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
		system("pause");
	}
}


__host__
void
gpuDestroyVars()
{
	hipFree(gpuptrElements);
	hipFree(gpuptrNodes);
	hipFree(gpuptrMulData);
	hipFree(gpuptr_xt);
	hipFree(gpuptr_vt);
	hipFree(gpuptr_extforces);
	hipFree(gpuptr_mass);
	hipFree(gpuptr_b);
	hipFree(gpuptrR);
	hipFree(gpuptrD);
	hipFree(gpuptrQ);
	hipFree(gpuptrVars);
}

__device__
void makeFU(float f0[12][BLOCK_SIZE], float R[3][3], float out[12])
{
	int ltid = threadIdx.x;
	float x[12];

	#pragma unroll 12
	for(int i=0;i<12;i++)
		x[i] = f0[i][ltid];

	#pragma unroll 4
	for(int i=0;i<4;i++)
		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			out[i*3 + j] = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
			out[i*3+j] += R[j][k] * x[i*3 + k];
		}		
}

__device__
void mulK(float x[12], float B[3][3][BLOCK_SIZE], float c1[BLOCK_SIZE], float c2[BLOCK_SIZE])
{
	int ltid = threadIdx.x;
	float temp[6];
	float temp2[6];
	float b[4][3];

	#pragma unroll 3
	for(int i=0;i<3;i++)
		#pragma unroll 3
		for(int j=0;j<3;j++)
			b[i][j] = B[i][j][ltid];
	
	b[3][0] = -b[0][0] -b[1][0] -b[2][0];
	b[3][1] = -b[0][1] -b[1][1] -b[2][1];
	b[3][2] = -b[0][2] -b[1][2] -b[2][2];

	float con1 = c1[ltid];
	float con2 = c2[ltid];
	float con3 = (con1 - con2)/2.0;

	temp[0] = b[0][0] * x[0] + b[1][0] * x[3] + b[2][0] * x[6] + b[3][0] * x[9];
	temp[1] = b[0][1] * x[1] + b[1][1] * x[4] + b[2][1] * x[7] + b[3][1] * x[10];
	temp[2] = b[0][2] * x[2] + b[1][2] * x[5] + b[2][2] * x[8] + b[3][2] * x[11];
	temp[3] = b[0][1] * x[0] + b[0][0] * x[1] + b[1][1] * x[3] + b[1][0] * x[4] + b[2][1] * x[6] + b[2][0] * x[7] + b[3][1] * x[9] + b[3][0] * x[10];
	temp[4] = b[0][2] * x[1] + b[0][1] * x[2] + b[1][2] * x[4] + b[1][1] * x[5] + b[2][2] * x[7] + b[2][1] * x[8] + b[3][2] * x[10] + b[3][1] * x[11];
	temp[5] = b[0][2] * x[0] + b[0][0] * x[2] + b[1][2] * x[3] + b[1][0] * x[5] + b[2][2] * x[6] + b[2][0] * x[8] + b[3][2] * x[9] + b[3][0] * x[11];

	temp2[0] = temp[0] * con1 + temp[1] * con2 + temp[2] * con2;
	temp2[1] = temp[0] * con2 + temp[1] * con1 + temp[2] * con2;
	temp2[2] = temp[0] * con2 + temp[1] * con2 + temp[2] * con1;
	temp2[3] = temp[3] * con3;
	temp2[4] = temp[4] * con3;
	temp2[5] = temp[5] * con3;

	x[0] = b[0][0] * temp2[0] + b[0][1] * temp2[3] + b[0][2] * temp2[5];
	x[1] = b[0][1] * temp2[1] + b[0][0] * temp2[3] + b[0][2] * temp2[4];
	x[2] = b[0][2] * temp2[2] + b[0][1] * temp2[4] + b[0][0] * temp2[5];

	x[3] = b[1][0] * temp2[0] + b[1][1] * temp2[3] + b[1][2] * temp2[5];
	x[4] = b[1][1] * temp2[1] + b[1][0] * temp2[3] + b[1][2] * temp2[4];
	x[5] = b[1][2] * temp2[2] + b[1][1] * temp2[4] + b[1][0] * temp2[5];

	x[6] = b[2][0] * temp2[0] + b[2][1] * temp2[3] + b[2][2] * temp2[5];
	x[7] = b[2][1] * temp2[1] + b[2][0] * temp2[3] + b[2][2] * temp2[4];
	x[8] = b[2][2] * temp2[2] + b[2][1] * temp2[4] + b[2][0] * temp2[5];

	x[9] = b[3][0] * temp2[0] + b[3][1] * temp2[3] + b[3][2] * temp2[5];
	x[10] = b[3][1] * temp2[1] + b[3][0] * temp2[3] + b[3][2] * temp2[4];
	x[11] = b[3][2] * temp2[2] + b[3][1] * temp2[4] + b[3][0] * temp2[5];

}

__device__
void makeRKRT(float B[3][3][BLOCK_SIZE], float c1[BLOCK_SIZE], float c2[BLOCK_SIZE], float R[3][3], float xt[12], float b[12])
{
	float temp[12];

	#pragma unroll 4
	for(int i=0;i<4;i++)
		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			temp[i*3 + j] = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
			temp[i*3+j] += R[k][j] * xt[i*3 + k]; //RT first
		}

	mulK(temp, B,c1,c2);

	#pragma unroll 4
	for(int i=0;i<4;i++)
		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			#pragma unroll 3
			for(int k=0;k<3;k++)
			b[i*3+j] -= R[j][k] * temp[i*3 + k];
		}

}

__device__
void mulSystem(GPUElement* elements, mulData* solverData, float* x)
{
	int bid = blockIdx.x;
	int ltid = threadIdx.x;

	GPUElement* t_ele = &(elements[bid]);
	mulData* t_solvedata = &(solverData[bid]);

	float nodes[12];
	float temp[12];
	float R[3][3];
	//float nodalmass = t_ele->nodalmass[ltid] * COEFFM;

	#pragma unroll 4
	for(int i=0;i<4;i++)
	{
		int index = t_ele->nodeindex[i][ltid];
		nodes[i * 3] = x[index * 3];
		nodes[i * 3 + 1] = x[index * 3 + 1];
		nodes[i * 3 + 2] = x[index * 3 + 2];
	}

	#pragma unroll 3
	for(int i=0;i<3;i++)
		#pragma unroll 3
		for(int j=0;j<3;j++)
			R[i][j] = t_solvedata->R[i][j][ltid];
	
	//rotate by x by RT first
	#pragma unroll 4
	for(int i=0;i<4;i++)
		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			temp[i*3 + j] = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
			temp[i*3+j] += R[k][j] * nodes[i*3 + k];
		}

	mulK(temp, t_ele->B, t_ele->c1, t_ele->c2);

	// rotate by R
	#pragma unroll 4
	for(int i=0;i<4;i++)
		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			float temp3 = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
				temp3 += R[j][k] * temp[i*3 + k];
			temp3 *= COEFFK;
			//temp3 += nodalmass * nodes[i*3 + j];
			t_solvedata->product[i*3 + j][ltid] = temp3;
		}
}

__device__
void dot(float*a, float*b, float* out, int n) 
{
	__shared__ float temp[DOT_BLOCK_SIZE];
	int index = threadIdx.x;
	int element = index;

	float tmp = 0;

	while(element < n)
	{
		tmp += a[element] * b[element];
		element += DOT_BLOCK_SIZE;
	}

	temp[index] = tmp;

	__syncthreads();


	int i = DOT_BLOCK_SIZE >> 1;
	while(i>0)
	{
		if(index < i)
			temp[index] += temp[index + i];
		__syncthreads();
		i>>=1;
	}

	if(index == 0)
		*out = temp[0];
}

//step 1
//precompute
__global__
void precompute(GPUElement* elements, mulData* solverData, float* xt, float* vt, float* extforces, int numelements)
{
	int tid = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	int bid = blockIdx.x;
	int ltid = threadIdx.x;

	if(tid < numelements)
	{
		GPUElement* t_ele = &(elements[bid]);
		mulData* t_solvedata = &(solverData[bid]);

		float nodes[12], b[12], R[3][3]={0}, D[3][3];
		int index[4];
			

		#pragma unroll 4
		for(int i=0;i<4;i++)
		{
			index[i] = t_ele->nodeindex[i][ltid];
			nodes[i * 3] = xt[index[i] * 3];
			nodes[i * 3 + 1] = xt[index[i] * 3 + 1];
			nodes[i * 3 + 2] = xt[index[i] * 3 + 2];
		}

		#pragma unroll 3
		for(int i=0;i<3;i++)
			#pragma unroll 3
			for(int j=0;j<3;j++)
				D[i][j] = t_ele->B[i][j][ltid];

		#pragma unroll 3
		for(int i=0;i<3;i++)
			#pragma unroll 3
			for(int j=0;j<3;j++)
					R[i][j] = (nodes[i] - nodes[9 + i]) * D[0][j] + 
							  (nodes[3 + i] - nodes[9 + i]) * D[1][j] + 
							  (nodes[6 + i] - nodes[9 + i]) * D[2][j];

		gpuComputePolarDecomposition(R);
	
		#pragma unroll 3
		for(int i=0;i<3;i++)
			#pragma unroll 3
			for(int j=0;j<3;j++)
				t_solvedata->R[i][j][ltid] =  R[i][j];

		makeFU(t_ele->f0,R,b);
	
		makeRKRT(t_ele->B,t_ele->c1,t_ele->c2, R, nodes, b);

		#pragma unroll 4
		for(int i=0;i<4;i++)
		{
			b[i * 3] += extforces[index[i] * 3];
			b[i * 3 + 1] += extforces[index[i] * 3 + 1];
			b[i * 3 + 2] += extforces[index[i] * 3 + 2];
		}

		//float nodalmass = t_ele->nodalmass[ltid];

		#pragma unroll 12
		for(int i=0;i<12;i++)
			t_solvedata->b[i][ltid] = b[i] * dt;// + nodalmass * vt[index[(i/3)] * 3 + (i%3)];

	}
}

//step 2
//precompute
__global__
void gatherB(GPUNode* nodes, mulData* solverData, float* b, float* mass, float* vt, char* allowed,int numnodes)
{
	int groupid = threadIdx.x % NODE_BLOCK_SIZE;// / NODE_THREADS;
	int grouptid = threadIdx.x / NODE_BLOCK_SIZE; //% NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_THREADS][NODE_BLOCK_SIZE][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[grouptid][groupid];
	
	if(nodeno < numnodes)
	{

		cache[grouptid][groupid][0] = 0;
		cache[grouptid][groupid][1] = 0;
		cache[grouptid][groupid][2] = 0;


		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][grouptid][groupid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][grouptid][groupid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][grouptid][groupid];

			cache[grouptid][groupid][0] += solverData[tetindex].b[nodeindex * 3][tetindex2];
			cache[grouptid][groupid][1] += solverData[tetindex].b[nodeindex * 3 + 1][tetindex2];
			cache[grouptid][groupid][2] += solverData[tetindex].b[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{
			b[nodeno * 3]     = cache[0][groupid][0] + cache[1][groupid][0] + mass[nodeno * 3] * vt[nodeno * 3];
			b[nodeno * 3 + 1] = cache[0][groupid][1] + cache[1][groupid][1] + mass[nodeno * 3 + 1] * vt[nodeno * 3 + 1];
			b[nodeno * 3 + 2] = cache[0][groupid][2] + cache[1][groupid][2] + mass[nodeno * 3 + 2] * vt[nodeno * 3 + 2];

			char bitsy = allowed[nodeno];
			if(bitsy & 1)
				vt[nodeno * 3] = 0;
			if(bitsy & 2)
				vt[nodeno * 3 + 1] = 0;
			if(bitsy & 4)
				vt[nodeno * 3 + 2] = 0;

		}
	}
}

//step 1
//init CG
// x = velocity
__global__
void
initAx(GPUElement* elements, mulData* solverData, float* x, int numelements)
{
	int tid = threadIdx.x + blockIdx.x * BLOCK_SIZE;

	if(tid < numelements)
	{
		mulSystem(elements, solverData, x);
	}
}

//step2
//init CG
__global__
void
initRandD(GPUNode* nodes, mulData* solverData, float* r, float* d, float* b, float* mass, float* vt, char* allowed, int numnodes)
{
	int groupid = threadIdx.x % NODE_BLOCK_SIZE;// / NODE_THREADS;
	int grouptid = threadIdx.x / NODE_BLOCK_SIZE; //% NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_THREADS][NODE_BLOCK_SIZE][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[grouptid][groupid];
	
	if(nodeno < numnodes)
	{

		cache[grouptid][groupid][0] = 0;
		cache[grouptid][groupid][1] = 0;
		cache[grouptid][groupid][2] = 0;

		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][grouptid][groupid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][grouptid][groupid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][grouptid][groupid];

			cache[grouptid][groupid][0] += solverData[tetindex].product[nodeindex * 3][tetindex2];
			cache[grouptid][groupid][1] += solverData[tetindex].product[nodeindex * 3 + 1][tetindex2];
			cache[grouptid][groupid][2] += solverData[tetindex].product[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{	
			char bitsy = allowed[nodeno];

			//r = b-Ax
			float r0 =  (bitsy & 1) ? 0 : (b[nodeno * 3] - (cache[0][groupid][0] + cache[1][groupid][0] + mass[nodeno * 3] * vt[nodeno * 3] * COEFFM));
			float r1 =  (bitsy & 2) ? 0 : (b[nodeno * 3 + 1] - (cache[0][groupid][1] + cache[1][groupid][1] + mass[nodeno * 3 + 1] * vt[nodeno * 3 + 1] * COEFFM));
			float r2 =  (bitsy & 4) ? 0 : (b[nodeno * 3 + 2] - (cache[0][groupid][2] + cache[1][groupid][2] + mass[nodeno * 3 + 2] * vt[nodeno * 3 + 2] * COEFFM));

			r[nodeno * 3] = r0;
			r[nodeno * 3 + 1] = r1;
			r[nodeno * 3 + 2] = r2;

			//d=r
			d[nodeno * 3] = r0;
			d[nodeno * 3 + 1] = r1;
			d[nodeno * 3 + 2] = r2;
		}
	}

}

//step3
//init CG
//1 block, BLOCK_SIZE threads
__global__
void
initDeltaVars(CGVars* vars, float* r, int numnodes)
{
	__shared__ float rr;
	dot(r, r, &rr, numnodes * 3);
	
	if(threadIdx.x == 0)
	{
		vars->deltaNew = rr;
		vars->delta0 = vars->deltaNew;
	}
}

//step 4
//CG loop
//q = Ad
__global__
void
makeQprod(GPUElement* elements, mulData* solverData, float* d, int numelements)
{
	int tid = threadIdx.x + blockIdx.x * BLOCK_SIZE;

	if(tid < numelements)
	{
		mulSystem(elements, solverData, d);
	}
}

//step 5
//CG loop
//q = Ad
__global__
void
gatherQprod(GPUNode* nodes, mulData* solverData, float* q, float* mass, float* d, char* allowed, int numnodes)
{
	int groupid = threadIdx.x % NODE_BLOCK_SIZE;// / NODE_THREADS;
	int grouptid = threadIdx.x / NODE_BLOCK_SIZE; //% NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_THREADS][NODE_BLOCK_SIZE][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[grouptid][groupid];
	
	if(nodeno < numnodes)
	{

		cache[grouptid][groupid][0] = 0;
		cache[grouptid][groupid][1] = 0;
		cache[grouptid][groupid][2] = 0;

		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][grouptid][groupid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][grouptid][groupid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][grouptid][groupid];

			cache[grouptid][groupid][0] += solverData[tetindex].product[nodeindex * 3][tetindex2];
			cache[grouptid][groupid][1] += solverData[tetindex].product[nodeindex * 3 + 1][tetindex2];
			cache[grouptid][groupid][2] += solverData[tetindex].product[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{
			char bitsy = allowed[nodeno];
			q[nodeno * 3]     = (bitsy & 1) ? 0 : (cache[0][groupid][0] + cache[1][groupid][0] + mass[nodeno * 3] * d[nodeno * 3] * COEFFM);
			q[nodeno * 3 + 1] = (bitsy & 2) ? 0 : (cache[0][groupid][1] + cache[1][groupid][1] + mass[nodeno * 3 + 1] * d[nodeno * 3 + 1] * COEFFM);
			q[nodeno * 3 + 2] = (bitsy & 4) ? 0 : (cache[0][groupid][2] + cache[1][groupid][2] + mass[nodeno * 3 + 2] * d[nodeno * 3 + 2] * COEFFM);
		}
	}

}

//step 6
//CG Loop
//make vars
//1 block, BLOCK_SIZE threads
__global__
void
makeVars(CGVars* vars, float* d, float* q, float* r, int numnodes)
{
	float dq, rq, qq;
	dot(d,q,&dq,numnodes * 3);
	dot(r,q,&rq,numnodes * 3);
	dot(q,q,&qq,numnodes * 3);

	__syncthreads();

	if(threadIdx.x == 0)
	{
		vars->alpha = vars->deltaNew / dq;
		vars->deltaOld = vars->deltaNew;

		//r.r = r'.r' - 2*alpha*(r'.q) + alpha * alpha * (q.q)
		vars->deltaNew = vars->deltaNew - (2 * vars->alpha) * rq + (vars->alpha * vars->alpha) * qq;
		vars->beta = vars->deltaNew / vars->deltaOld;
	}
}

//step 7
//CG Loop
//make x, r, d
//x = velocity
__global__
void
makeXRandD(CGVars* vars, float *x, float* r, float* d, float* q, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x * VECTOR_BLOCK_SIZE;
	if(tid < numnodes)
	{
		float alpha = vars->alpha;
		float beta = vars->beta;

		x[tid * 3] = x[tid * 3] + alpha * d[tid * 3];
		x[tid * 3 + 1] = x[tid * 3 + 1] + alpha * d[tid * 3 + 1];
		x[tid * 3 + 2] = x[tid * 3 + 2] + alpha * d[tid * 3 + 2];

		r[tid * 3] = r[tid * 3] - alpha * q[tid * 3];
		r[tid * 3 + 1] = r[tid * 3 + 1] - alpha * q[tid * 3 + 1];
		r[tid * 3 + 2] = r[tid * 3 + 2] - alpha * q[tid * 3 + 2];

		d[tid * 3] = r[tid * 3] + beta * d[tid * 3];
		d[tid * 3 + 1] = r[tid * 3 + 1] + beta * d[tid * 3 + 1];
		d[tid * 3 + 2] = r[tid * 3 + 2] + beta * d[tid * 3 + 2];
	}
} 

//step 8
//make x(t+1)
__global__
void
integrate(float *x, float* v, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x * VECTOR_BLOCK_SIZE;
	if(tid < numnodes)
	{
		x[tid * 3] = x[tid * 3] + dt * v[tid * 3];
		x[tid * 3 + 1] = x[tid * 3 + 1] + dt * v[tid * 3 + 1];
		x[tid * 3 + 2] = x[tid * 3 + 2] + dt * v[tid * 3 + 2];
	}
}

__host__
void
gpuTimeStep(int numelements, int numnodes)
{
	const int num_blocks_ele = (numelements/BLOCK_SIZE) + 1;
	const int num_blocks_node = (numnodes/NODE_BLOCK_SIZE) + 1;
	const int num_blocks_vec = (numnodes/VECTOR_BLOCK_SIZE) + 1;

	hipError_t error;

	printf("Started\n");
	
	precompute<<<num_blocks_ele, BLOCK_SIZE>>>(gpuptrElements, gpuptrMulData, gpuptr_xt, gpuptr_vt, gpuptr_extforces, numelements);
	
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("1");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}


	gatherB<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptr_b, gpuptr_mass, gpuptr_vt, gpuptr_allowed, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("2");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	initAx<<<num_blocks_ele, BLOCK_SIZE>>>(gpuptrElements, gpuptrMulData, gpuptr_vt, numelements);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("3");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	initRandD<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptrR, gpuptrD, gpuptr_b, gpuptr_mass, gpuptr_vt,  gpuptr_allowed,numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("4");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	initDeltaVars<<<1, DOT_BLOCK_SIZE>>>(gpuptrVars, gpuptrR, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("5");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	int i=0;

	CGVars vars;
	hipMemcpy(&vars, gpuptrVars, sizeof(CGVars), hipMemcpyDeviceToHost);

	printf("Loop Started");

	while(i < MAX_ITER && vars.deltaNew > (EPSIL * EPSIL) * vars.delta0)
	{
		makeQprod<<<num_blocks_ele, BLOCK_SIZE>>>(gpuptrElements, gpuptrMulData, gpuptrD, numelements);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("6");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		gatherQprod<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptrQ, gpuptr_mass, gpuptrD, gpuptr_allowed,numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("7");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		makeVars<<<1, DOT_BLOCK_SIZE>>>(gpuptrVars, gpuptrD, gpuptrQ, gpuptrR, numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("8");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		makeXRandD<<<num_blocks_vec, VECTOR_BLOCK_SIZE>>>(gpuptrVars, gpuptr_vt, gpuptrR, gpuptrD, gpuptrQ, numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("9");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		hipMemcpy(&vars, gpuptrVars, sizeof(CGVars), hipMemcpyDeviceToHost);
		i++;

	}

	printf("Loop Ended: %d\n", i);

	integrate<<<num_blocks_vec, VECTOR_BLOCK_SIZE>>>(gpuptr_xt, gpuptr_vt, numnodes);
}


#endif














//EXTRAS

//completely mat free version.

/*

__device__
void mulSystemGather(GPUNode* nodes, mulData* solverData, float* x, int numnodes)
{
	int groupid = threadIdx.x / NODE_THREADS;
	int grouptid = threadIdx.x % NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_BLOCK_SIZE][NODE_THREADS][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[groupid][grouptid];
	
	if(nodeno < numnodes)
	{

		cache[groupid][grouptid][0] = 0;
		cache[groupid][grouptid][1] = 0;
		cache[groupid][grouptid][2] = 0;

		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][groupid][grouptid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][groupid][grouptid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][groupid][grouptid];

			cache[groupid][grouptid][0] += solverData[tetindex].product[nodeindex * 3][tetindex2];
			cache[groupid][grouptid][1] += solverData[tetindex].product[nodeindex * 3 + 1][tetindex2];
			cache[groupid][grouptid][2] += solverData[tetindex].product[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{
			x[nodeno * 3]     = cache[groupid][0][0] + cache[groupid][1][0] + cache[groupid][2][0] + cache[groupid][3][0];
			x[nodeno * 3 + 1] = cache[groupid][0][1] + cache[groupid][1][1] + cache[groupid][2][1] + cache[groupid][3][1];
			x[nodeno * 3 + 2] = cache[groupid][0][2] + cache[groupid][1][2] + cache[groupid][2][2] + cache[groupid][3][2];
		}
	}

}



__device__
void mulK(float x[12], float B[3][3][BLOCK_SIZE], float c1[BLOCK_SIZE], float c2[BLOCK_SIZE])
{
	int ltid = threadIdx.x;
	float temp[6];
	float temp2[6];
	float b[3][3];

	for(int i=0;i<3;i++)
		for(int j=0;j<3;j++)
			b[i][j] = B[i][j][ltid];
	
	float con1 = c1[ltid] * COEFFK;
	float con2 = c2[ltid] * COEFFK;
	float con3 = (con1 - con2)/2.0;

	float b4 = -b[0][0] -b[1][0] -b[2][0]; 
	float c4 = -b[0][1] -b[1][1] -b[2][1]; 
	float d4 = -b[0][2] -b[1][2] -b[2][2]; 

	temp[0] = b[0][0] * x[0] + b[1][0] * x[3] + b[2][0] * x[6] + b4 * x[9];
	temp[1] = b[0][1] * x[1] + b[1][1] * x[4] + b[2][1] * x[7] + c4 * x[10];
	temp[2] = b[0][2] * x[2] + b[1][2] * x[5] + b[2][2] * x[8] + d4 * x[11];
	temp[3] = b[0][1] * x[0] + b[0][0] * x[1] + b[1][1] * x[3] + b[1][0] * x[4] + b[2][1] * x[6] + b[2][0] * x[7] + c4 * x[9] + b4 * x[10];
	temp[4] = b[0][2] * x[1] + b[0][1] * x[2] + b[1][2] * x[4] + b[1][1] * x[5] + b[2][2] * x[7] + b[2][1] * x[8] + d4 * x[10] + c4 * x[11];
	temp[5] = b[0][2] * x[0] + b[0][0] * x[2] + b[1][2] * x[3] + b[1][0] * x[5] + b[2][2] * x[6] + b[2][0] * x[8] + d4 * x[9] + b4 * x[11];

	temp2[0] = temp[0] * con1 + temp[1] * con2 + temp[2] * con2;
	temp2[1] = temp[0] * con2 + temp[1] * con1 + temp[2] * con2;
	temp2[2] = temp[0] * con2 + temp[1] * con2 + temp[2] * con1;
	temp2[3] = temp[3] * con3;
	temp2[4] = temp[4] * con3;
	temp2[5] = temp[5] * con3;

	x[0] = b[0][0] * temp2[0] + b[0][1] * temp2[3] + b[0][2] * temp2[5];
	x[1] = b[0][1] * temp2[1] + b[0][0] * temp2[3] + b[0][2] * temp2[4];
	x[2] = b[0][2] * temp2[2] + b[0][1] * temp2[4] + b[0][0] * temp2[5];

	x[3] = b[1][0] * temp2[0] + b[1][1] * temp2[3] + b[1][2] * temp2[5];
	x[4] = b[1][1] * temp2[1] + b[1][0] * temp2[3] + b[1][2] * temp2[4];
	x[5] = b[1][2] * temp2[2] + b[1][1] * temp2[4] + b[1][0] * temp2[5];

	x[6] = b[2][0] * temp2[0] + b[2][1] * temp2[3] + b[2][2] * temp2[5];
	x[7] = b[2][1] * temp2[1] + b[2][0] * temp2[3] + b[2][2] * temp2[4];
	x[8] = b[2][2] * temp2[2] + b[2][1] * temp2[4] + b[2][0] * temp2[5];

	x[9] = b4 * temp2[0] + c4 * temp2[3] + d4 * temp2[5];
	x[10] = c4 * temp2[1] + b4 * temp2[3] + d4 * temp2[4];
	x[11] = d4 * temp2[2] + c4 * temp2[4] + b4 * temp2[5];
}


__device__
void mulRKRT(float x[12], float R[3][3], float B[3][3][BLOCK_SIZE], float c1[BLOCK_SIZE], float c2[BLOCK_SIZE], float nodalmass)
{
	float temp[12];
	for(int i=0;i<4;i++)
		for(int j=0;j<3;j++)
		{
			temp[i*3 + j] = 0;;
			for(int k=0;k<3;k++)
			temp[i*3+j] += R[k][j] * x[i*3 + k]; //R[j][k] but RT so R[k][j]
		}

	mulK(temp, B, c1, c2);
	
	for(int i=0;i<4;i++)
		for(int j=0;j<3;j++)
		{
			temp[i*3 + j] = 0;
			for(int k=0;k<3;k++)
			temp[i*3+j] += R[j][k] * x[i*3 + k]; 
		}

	for(int i=0;i<12;i++)
	{
		x[i] = temp[i] + nodalmass * x[i]; 
	}

}


*/