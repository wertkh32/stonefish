#include "hip/hip_runtime.h"
#pragma once
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "defines.h"
#include "GPUDataStructs.cuh"
#include "GPUPolarDecompose.cu"

//#define BLOCK_SIZE 512

#define ALPHA 0.3
#define BETA 0.1

#define MAX_ITER 20
#define EPSIL 0.01

__constant__ float COEFFK, COEFFM, dt;

GPUElement* gpuptrElements;
GPUNode*   gpuptrNodes;
mulData*	gpuptrMulData;
float*   gpuptr_xt;//dynamic
float*   gpuptr_vt;//dynamic
float*	 gpuptr_extforces;//dynamic
float*	 gpuptr_b;//dynamic

//for CG
float* gpuptrR;
float* gpuptrD;
float* gpuptrQ;
CGVars* gpuptrVars;

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
		system("pause");
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__host__
void
gpuInitVars(int numele, int numnodes)
{
	int numblocksperele = (numele / BLOCK_SIZE) + 1;
	int numblockpernode = (numnodes / NODE_BLOCK_SIZE) + 1;

	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	HANDLE_ERROR( hipMalloc(&gpuptrElements, numblocksperele * sizeof(GPUElement)) );
	HANDLE_ERROR( hipMalloc(&gpuptrMulData, numblocksperele * sizeof(mulData)) );
	HANDLE_ERROR( hipMalloc(&gpuptrNodes, numblockpernode * sizeof(GPUNode)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_xt, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_vt, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_extforces, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_b, numnodes * 3 * sizeof(float)) );

	HANDLE_ERROR( hipMalloc(&gpuptrR, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptrD, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptrQ, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptrVars, sizeof(CGVars)) );

	float coeffK = dt * BETA + dt * dt, coeffM = 1 + dt * ALPHA;
	float dt = 1.0/FPS;

	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL("COEFFK"), &coeffK, sizeof(float)) );
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL("COEFFM"), &coeffM, sizeof(float)) );
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL("dt"), &dt, sizeof(float)) );

		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
			system("pause");
		}

	//testing
	//float F[3][3] = { {1,2,3}, {3,2,1}, {1,3,2} };
	//float R[3][3];
	//gpuComputePolarDecomposition(F,R);

	//for(int i=0;i<3;i++, printf("\n"))
	//	for(int j=0; j<3; j++)
	//		printf("%f ", R[i][j]);
	//system("pause");
}

__host__
void
gpuUploadVars(GPUElement* gpuElements, GPUNode* gpuNodes,float* xt, 
			  float* vt, float* extforces, int numnodes, int numelements)
{
	int numblocksperele = (numelements / BLOCK_SIZE) + 1;
	int numblockpernode = (numnodes / NODE_BLOCK_SIZE) + 1;

	HANDLE_ERROR( hipMemcpy(gpuptrElements, gpuElements, numblocksperele * sizeof(GPUElement), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptrNodes, gpuNodes, numblockpernode * sizeof(GPUNode), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_xt, xt, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_vt, vt, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_extforces, extforces, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );

		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
			system("pause");
		}
}

__host__
void
gpuDownloadVars(float* xt, int numnodes)
{
	hipMemcpy(xt, gpuptr_xt, numnodes * 3 * sizeof(float), hipMemcpyDeviceToHost);
}

__host__
void
gpuUploadExtForces(float* extforces, int numnodes)
{
	hipMemcpy(gpuptr_extforces, extforces, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
		system("pause");
	}
}


__host__
void
gpuDestroyVars()
{
	hipFree(gpuptrElements);
	hipFree(gpuptrNodes);
	hipFree(gpuptrMulData);
	hipFree(gpuptr_xt);
	hipFree(gpuptr_vt);
	hipFree(gpuptr_extforces);
	hipFree(gpuptr_b);
	hipFree(gpuptrR);
	hipFree(gpuptrD);
	hipFree(gpuptrQ);
	hipFree(gpuptrVars);
}

__device__
void makeFU(float x0[12][BLOCK_SIZE], float mat[12][12], float R[3][3], float out[12])
{
	int ltid = threadIdx.x;
	float kx[12];
	float x[12];

	#pragma unroll 12
	for(int i=0;i<12;i++)
		x[i] = x0[i][ltid];
	
	for(int i=0;i<12;i++)
	{
		kx[i] = 0;
		for(int j=0;j<12;j++)
			kx[i] += mat[i][j] * x[j];
	}

	for(int i=0;i<4;i++)
		for(int j=0;j<3;j++)
		{
			out[i*3 + j] = 0;
			for(int k=0;k<3;k++)
			out[i*3+j] += R[j][k] * kx[i*3 + k];
		}		
				
}

__device__
void makeRKRT(float mat[12][12], float R[3][3])
{
	float temp[12][12];
	//int ltid = threadIdx.x;

	for(int i=0;i<4;i++)
	{
		for(int j=i;j<4;j++)
		{
			for(int a=0;a<3;a++)
				for(int b=0;b<3;b++)
				{
					temp[a + i * 3][b + j * 3]= 0.0f;
						
					for(int c=0;c<3;c++)
						temp[a + i * 3][b + j * 3] += R[a][c] * mat[c + i * 3][b + j * 3];
				}
		}
	}

	for(int i=0;i<4;i++)
	{
		for(int j=i;j<4;j++)
		{
			for(int a=0;a<3;a++)
				for(int b=0;b<3;b++)
				{
					mat[a + i * 3][b + j * 3] = 0.0f;
					for(int c=0;c<3;c++)
						mat[a + i * 3][b + j * 3] += temp[a + i * 3][c + j * 3] * R[b][c]; //R(c, b) but its RT so, R(b , c)
				}
		}
	}



	//lower triangle
	for(int i=1;i<4;i++)
		for(int j=0;j<i;j++)
		{
			for(int a=0;a<3;a++)
				for(int b=0;b<3;b++)
						mat[a + i * 3][b + j * 3] = mat[b + j * 3][a + i * 3];
		}

}

__device__
void mulSystem(GPUElement* elements, mulData* solverData, float* x)
{
	int bid = blockIdx.x;
	int ltid = threadIdx.x;

	GPUElement* t_ele = &(elements[bid]);
	mulData* t_solvedata = &(solverData[bid]);

	float nodes[12];

	#pragma unroll 4
	for(int i=0;i<4;i++)
	{
		int index = t_ele->nodeindex[i][ltid];
		nodes[i * 3] = x[index * 3];
		nodes[i * 3 + 1] = x[index * 3 + 1];
		nodes[i * 3 + 2] = x[index * 3 + 2];
	}

	for(int i=0;i<12;i++)
	{
		float temp = 0;
		#pragma unroll 12
		for(int j=0;j<12;j++)
			 temp += t_solvedata->system[i][j][ltid] * nodes[j];
		t_solvedata->product[i][ltid] = temp;
	}
}

__device__
void mulSystemGather(GPUNode* nodes, mulData* solverData, float* x, int numnodes)
{
	int groupid = threadIdx.x / NODE_THREADS;
	int grouptid = threadIdx.x % NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_BLOCK_SIZE][NODE_THREADS][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[groupid][grouptid];
	
	if(nodeno < numnodes)
	{

		cache[groupid][grouptid][0] = 0;
		cache[groupid][grouptid][1] = 0;
		cache[groupid][grouptid][2] = 0;

		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][groupid][grouptid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][groupid][grouptid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][groupid][grouptid];

			cache[groupid][grouptid][0] += solverData[tetindex].product[nodeindex * 3][tetindex2];
			cache[groupid][grouptid][1] += solverData[tetindex].product[nodeindex * 3 + 1][tetindex2];
			cache[groupid][grouptid][2] += solverData[tetindex].product[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{
			x[nodeno * 3]     = cache[groupid][0][0] + cache[groupid][1][0] + cache[groupid][2][0] + cache[groupid][3][0];
			x[nodeno * 3 + 1] = cache[groupid][0][1] + cache[groupid][1][1] + cache[groupid][2][1] + cache[groupid][3][1];
			x[nodeno * 3 + 2] = cache[groupid][0][2] + cache[groupid][1][2] + cache[groupid][2][2] + cache[groupid][3][2];
		}
	}

}

__device__
void dot(float*a, float*b, float* out, int n) 
{
	__shared__ float temp[DOT_BLOCK_SIZE];
	int index = threadIdx.x;
	int element = index;

	float tmp = 0;

	while(element < n)
	{
		tmp += a[element] * b[element];
		element += DOT_BLOCK_SIZE;
	}

	temp[index] = tmp;

	__syncthreads();


	int i = DOT_BLOCK_SIZE >> 1;
	while(i>0)
	{
		if(index < i)
			temp[index] += temp[index + i];
		__syncthreads();
		i>>=1;
	}

	if(index == 0)
		*out = temp[0];
}

//step 1
//precompute
__global__
void precompute(GPUElement* elements, mulData* solverData, float* xt, float* vt, float* extforces, int numelements)
{
	int tid = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	int bid = blockIdx.x;
	int ltid = threadIdx.x;

	if(tid < numelements)
	{
		GPUElement* t_ele = &(elements[bid]);
		mulData* t_solvedata = &(solverData[bid]);

		float nodalmass = t_ele->nodalmass[ltid];

		float nodes[12], b[12], R[3][3]={0};// = {{1,0,0},{0,1,0},{0,0,1}};
		float K[12][12];
		int index[4];

		#pragma unroll 4
		for(int i=0;i<4;i++)
			index[i] = t_ele->nodeindex[i][ltid];

		#pragma unroll 4
		for(int i=0;i<4;i++)
		{
			nodes[i * 3] = xt[index[i] * 3];
			nodes[i * 3 + 1] = xt[index[i] * 3 + 1];
			nodes[i * 3 + 2] = xt[index[i] * 3 + 2];
		}

		for(int i=0;i<3;i++)
			for(int j=0;j<3;j++)
				for(int k=0;k<3;k++)
					R[i][j] += (nodes[k*3 + i] - nodes[9 + i]) * t_ele->undefShapeMatInv[k][j][ltid];

		gpuComputePolarDecomposition(R,R);


		for(int i=0;i<12;i++)
			for(int j=0;j<12;j++)
			{
				K[i][j] = t_ele->unwarpK[i][j][ltid];
			}
	
		makeFU(t_ele->x0,K,R,b);
	
		makeRKRT(K, R);


		for(int i=0;i<12;i++)
			for(int j=0;j<12;j++)
				b[i] -= K[i][j] * nodes[j];

		#pragma unroll 4
		for(int i=0;i<4;i++)
		{
			b[i * 3] += extforces[index[i] * 3];
			b[i * 3 + 1] += extforces[index[i] * 3 + 1];
			b[i * 3 + 2] += extforces[index[i] * 3 + 2];
		}

		//final system matrix
		for(int i=0;i<12;i++)
			for(int j=0;j<12;j++)
			{
				K[i][j] *= COEFFK;
				if(i==j)
					K[i][i] += COEFFM * nodalmass;
			}

		#pragma unroll 12
		for(int i=0;i<12;i++)
			t_solvedata->b[i][ltid] = b[i] * dt + nodalmass * vt[t_ele->nodeindex[(i/3)][ltid] * 3 + (i%3)];

		for(int i=0;i<12;i++)
			for(int j=0;j<12;j++)
				t_solvedata->system[i][j][ltid] =  K[i][j];
	}
}

//step 2
//precompute
__global__
void gatherB(GPUNode* nodes, mulData* solverData, float* b, int numnodes)
{
	int groupid = threadIdx.x / NODE_THREADS;
	int grouptid = threadIdx.x % NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_BLOCK_SIZE][NODE_THREADS][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[groupid][grouptid];
	
	if(nodeno < numnodes)
	{

		cache[groupid][grouptid][0] = 0;
		cache[groupid][grouptid][1] = 0;
		cache[groupid][grouptid][2] = 0;

		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][groupid][grouptid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][groupid][grouptid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][groupid][grouptid];

			cache[groupid][grouptid][0] += solverData[tetindex].b[nodeindex * 3][tetindex2];
			cache[groupid][grouptid][1] += solverData[tetindex].b[nodeindex * 3 + 1][tetindex2];
			cache[groupid][grouptid][2] += solverData[tetindex].b[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{
			b[nodeno * 3]     = cache[groupid][0][0] + cache[groupid][1][0] + cache[groupid][2][0] + cache[groupid][3][0];
			b[nodeno * 3 + 1] = cache[groupid][0][1] + cache[groupid][1][1] + cache[groupid][2][1] + cache[groupid][3][1];
			b[nodeno * 3 + 2] = cache[groupid][0][2] + cache[groupid][1][2] + cache[groupid][2][2] + cache[groupid][3][2];
		}
	}
}

//step 1
//init CG
// x = velocity
__global__
void
initAx(GPUElement* elements, mulData* solverData, float* x, int numelements)
{
	int tid = threadIdx.x + blockIdx.x * BLOCK_SIZE;

	if(tid < numelements)
	{
		mulSystem(elements, solverData, x);
	}
}

//step2
//init CG
__global__
void
initRandD(GPUNode* nodes, mulData* solverData, float* r, float* d, float* b, int numnodes)
{
	int groupid = threadIdx.x / NODE_THREADS;
	int grouptid = threadIdx.x % NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_BLOCK_SIZE][NODE_THREADS][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[groupid][grouptid];
	
	if(nodeno < numnodes)
	{

		cache[groupid][grouptid][0] = 0;
		cache[groupid][grouptid][1] = 0;
		cache[groupid][grouptid][2] = 0;

		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][groupid][grouptid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][groupid][grouptid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][groupid][grouptid];

			cache[groupid][grouptid][0] += solverData[tetindex].product[nodeindex * 3][tetindex2];
			cache[groupid][grouptid][1] += solverData[tetindex].product[nodeindex * 3 + 1][tetindex2];
			cache[groupid][grouptid][2] += solverData[tetindex].product[nodeindex * 3 + 2][tetindex2];
		}
	}
	
	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{
			r[nodeno * 3]     = cache[groupid][0][0] + cache[groupid][1][0] + cache[groupid][2][0] + cache[groupid][3][0];
			r[nodeno * 3 + 1] = cache[groupid][0][1] + cache[groupid][1][1] + cache[groupid][2][1] + cache[groupid][3][1];
			r[nodeno * 3 + 2] = cache[groupid][0][2] + cache[groupid][1][2] + cache[groupid][2][2] + cache[groupid][3][2];
		
			//r = b-Ax
			r[nodeno * 3] = b[nodeno * 3] - r[nodeno * 3];
			r[nodeno * 3 + 1] = b[nodeno * 3 + 1] - r[nodeno * 3 + 1];
			r[nodeno * 3 + 2] = b[nodeno * 3 + 2] - r[nodeno * 3 + 2];

			//d=r
			d[nodeno * 3] = r[nodeno * 3];
			d[nodeno * 3 + 1] = r[nodeno * 3 + 1];
			d[nodeno * 3 + 2] = r[nodeno * 3 + 2];
		}
	}

}

//step3
//init CG
//1 block, BLOCK_SIZE threads
__global__
void
initDeltaVars(CGVars* vars, float* r, int numnodes)
{
	__shared__ float rr;
	dot(r, r, &rr, numnodes * 3);
	
	if(threadIdx.x == 0)
	{
		vars->deltaNew = rr;
		vars->delta0 = vars->deltaNew;
	}
}

//step 4
//CG loop
//q = Ad
__global__
void
makeQprod(GPUElement* elements, mulData* solverData, float* d, int numelements)
{
	int tid = threadIdx.x + blockIdx.x * BLOCK_SIZE;

	if(tid < numelements)
	{
		mulSystem(elements, solverData, d);
	}
}

//step 5
//CG loop
//q = Ad
__global__
void
gatherQprod(GPUNode* nodes, mulData* solverData, float* q, int numnodes)
{
		mulSystemGather(nodes, solverData, q, numnodes);

}

//step 6
//CG Loop
//make vars
//1 block, BLOCK_SIZE threads
__global__
void
makeVars(CGVars* vars, float* d, float* q, float* r, int numnodes)
{
	float dq, rq, qq;
	dot(d,q,&dq,numnodes * 3);
	dot(r,q,&rq,numnodes * 3);
	dot(q,q,&qq,numnodes * 3);

	__syncthreads();

	if(threadIdx.x == 0)
	{
		vars->alpha = vars->deltaNew / dq;
		vars->deltaOld = vars->deltaNew;

		//r.r = r'.r' - 2*alpha*(r'.q) + alpha * alpha * (q.q)
		vars->deltaNew = vars->deltaNew - (2 * vars->alpha) * rq + (vars->alpha * vars->alpha) * qq;
		vars->beta = vars->deltaNew / vars->deltaOld;
	}
}

//step 7
//CG Loop
//make x, r, d
//x = velocity
__global__
void
makeXRandD(CGVars* vars, float *x, float* r, float* d, float* q, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x * VECTOR_BLOCK_SIZE;
	if(tid < numnodes)
	{
		float alpha = vars->alpha;
		float beta = vars->beta;

		x[tid * 3] = x[tid * 3] + alpha * d[tid * 3];
		x[tid * 3 + 1] = x[tid * 3 + 1] + alpha * d[tid * 3 + 1];
		x[tid * 3 + 2] = x[tid * 3 + 2] + alpha * d[tid * 3 + 2];

		r[tid * 3] = r[tid * 3] - alpha * q[tid * 3];
		r[tid * 3 + 1] = r[tid * 3 + 1] - alpha * q[tid * 3 + 1];
		r[tid * 3 + 2] = r[tid * 3 + 2] - alpha * q[tid * 3 + 2];

		d[tid * 3] = r[tid * 3] + beta * d[tid * 3];
		d[tid * 3 + 1] = r[tid * 3 + 1] + beta * d[tid * 3 + 1];
		d[tid * 3 + 2] = r[tid * 3 + 2] + beta * d[tid * 3 + 2];
	}
} 

//step 8
//make x(t+1)
__global__
void
integrate(float *x, float* v, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x * VECTOR_BLOCK_SIZE;
	if(tid < numnodes)
	{
		x[tid * 3] = x[tid * 3] + dt * v[tid * 3];
		x[tid * 3 + 1] = x[tid * 3 + 1] + dt * v[tid * 3 + 1];
		x[tid * 3 + 2] = x[tid * 3 + 2] + dt * v[tid * 3 + 2];
	}
}

__host__
void
gpuTimeStep(int numelements, int numnodes)
{
	const int num_blocks_ele = (numelements/BLOCK_SIZE) + 1;
	const int num_blocks_node = (numnodes/NODE_BLOCK_SIZE) + 1;
	const int num_blocks_vec = (numnodes/VECTOR_BLOCK_SIZE) + 1;

	hipError_t error;

	printf("Started\n");
	
	precompute<<<num_blocks_ele, BLOCK_SIZE>>>(gpuptrElements, gpuptrMulData, gpuptr_xt, gpuptr_vt, gpuptr_extforces, numelements);
	
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("1");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}


	gatherB<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptr_b, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("2");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	initAx<<<num_blocks_ele, BLOCK_SIZE>>>(gpuptrElements, gpuptrMulData, gpuptr_vt, numelements);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("3");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	initRandD<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptrR, gpuptrD, gpuptr_b, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("4");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	initDeltaVars<<<1, DOT_BLOCK_SIZE>>>(gpuptrVars, gpuptrR, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("5");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	int i=0;

	CGVars vars;
	hipMemcpy(&vars, gpuptrVars, sizeof(CGVars), hipMemcpyDeviceToHost);

	printf("Loop Started");

	while(i < MAX_ITER && vars.deltaNew > (EPSIL * EPSIL) * vars.delta0)
	{
		makeQprod<<<num_blocks_ele, BLOCK_SIZE>>>(gpuptrElements, gpuptrMulData, gpuptrD, numelements);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("6");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		gatherQprod<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptrQ, numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("7");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		makeVars<<<1, DOT_BLOCK_SIZE>>>(gpuptrVars, gpuptrD, gpuptrQ, gpuptrR, numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("8");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		makeXRandD<<<num_blocks_vec, VECTOR_BLOCK_SIZE>>>(gpuptrVars, gpuptr_vt, gpuptrR, gpuptrD, gpuptrQ, numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("9");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		hipMemcpy(&vars, gpuptrVars, sizeof(CGVars), hipMemcpyDeviceToHost);
		i++;

	}

	printf("Loop Ended: %d\n", i);

	integrate<<<num_blocks_vec, VECTOR_BLOCK_SIZE>>>(gpuptr_xt, gpuptr_vt, numnodes);
}



















//EXTRAS

//completely mat free version.

/*
__device__
void mulK(float x[12], float B[3][3][BLOCK_SIZE], float c1[BLOCK_SIZE], float c2[BLOCK_SIZE])
{
	int ltid = threadIdx.x;
	float temp[6];
	float temp2[6];
	float b[3][3];

	for(int i=0;i<3;i++)
		for(int j=0;j<3;j++)
			b[i][j] = B[i][j][ltid];
	
	float con1 = c1[ltid];
	float con2 = c2[ltid];
	float con3 = (con1 - con2)/2.0;

	float b4 = -b[0][0] -b[1][0] -b[2][0]; 
	float c4 = -b[0][1] -b[1][1] -b[2][1]; 
	float d4 = -b[0][2] -b[1][2] -b[2][2]; 

	temp[0] = b[0][0] * x[0] + b[1][0] * x[3] + b[2][0] * x[6] + b4 * x[9];
	temp[1] = b[0][1] * x[1] + b[1][1] * x[4] + b[2][1] * x[7] + c4 * x[10];
	temp[2] = b[0][2] * x[2] + b[1][2] * x[5] + b[2][2] * x[8] + d4 * x[11];
	temp[3] = b[0][1] * x[0] + b[0][0] * x[1] + b[1][1] * x[3] + b[1][0] * x[4] + b[2][1] * x[6] + b[2][0] * x[7] + c4 * x[9] + b4 * x[10];
	temp[4] = b[0][2] * x[1] + b[0][1] * x[2] + b[1][2] * x[4] + b[1][1] * x[5] + b[2][2] * x[7] + b[2][1] * x[8] + d4 * x[10] + c4 * x[11];
	temp[5] = b[0][2] * x[0] + b[0][0] * x[2] + b[1][2] * x[3] + b[1][0] * x[5] + b[2][2] * x[6] + b[2][0] * x[8] + d4 * x[9] + b4 * x[11];

	temp2[0] = temp[0] * con1 + temp[1] * con2 + temp[2] * con2;
	temp2[1] = temp[0] * con2 + temp[1] * con1 + temp[2] * con2;
	temp2[2] = temp[0] * con2 + temp[1] * con2 + temp[2] * con1;
	temp2[3] = temp[3] * con3;
	temp2[4] = temp[4] * con3;
	temp2[5] = temp[5] * con3;

	x[0] = b[0][0] * temp2[0] + b[0][1] * temp2[3] + b[0][2] * temp2[5];
	x[1] = b[0][1] * temp2[1] + b[0][0] * temp2[3] + b[0][2] * temp2[4];
	x[2] = b[0][2] * temp2[2] + b[0][1] * temp2[4] + b[0][0] * temp2[5];

	x[3] = b[1][0] * temp2[0] + b[1][1] * temp2[3] + b[1][2] * temp2[5];
	x[4] = b[1][1] * temp2[1] + b[1][0] * temp2[3] + b[1][2] * temp2[4];
	x[5] = b[1][2] * temp2[2] + b[1][1] * temp2[4] + b[1][0] * temp2[5];

	x[6] = b[2][0] * temp2[0] + b[2][1] * temp2[3] + b[2][2] * temp2[5];
	x[7] = b[2][1] * temp2[1] + b[2][0] * temp2[3] + b[2][2] * temp2[4];
	x[8] = b[2][2] * temp2[2] + b[2][1] * temp2[4] + b[2][0] * temp2[5];

	x[9] = b4 * temp2[0] + c4 * temp2[3] + d4 * temp2[5];
	x[10] = c4 * temp2[1] + b4 * temp2[3] + d4 * temp2[4];
	x[11] = d4 * temp2[2] + c4 * temp2[4] + b4 * temp2[5];
}
*/