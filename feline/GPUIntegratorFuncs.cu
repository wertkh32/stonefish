#include "hip/hip_runtime.h"
#pragma once
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "defines.h"
#include "GPUDataStructs.cuh"
#include "GPUPolarDecompose.cu"

//#define BLOCK_SIZE 512

#define ALPHA 0.3
#define BETA 0.1

#define MAX_ITER 20
#define EPSIL 0.01

__constant__ float COEFFK, COEFFM, dt;

GPUElement* gpuptrElements;
GPUNode*   gpuptrNodes;
mulData*	gpuptrMulData;
float*   gpuptr_xt;//dynamic
float*   gpuptr_vt;//dynamic
float*	 gpuptr_extforces;//dynamic
float*	 gpuptr_b;//dynamic

//for CG
float* gpuptrR;
float* gpuptrD;
float* gpuptrQ;
CGVars* gpuptrVars;

__host__
void
gpuInitVars(int numele, int numnodes)
{
	int numblocksperele = (numele / BLOCK_SIZE) + 1;
	int numblockpernode = (numnodes / NODE_BLOCK_SIZE) + 1;

	hipMalloc(&gpuptrElements, numblocksperele * sizeof(GPUElement));
	hipMalloc(&gpuptrMulData, numblocksperele * sizeof(mulData));
	hipMalloc(&gpuptrNodes, numblockpernode * sizeof(GPUNode));
	hipMalloc(&gpuptr_xt, numnodes * 3 * sizeof(float));
	hipMalloc(&gpuptr_vt, numnodes * 3 * sizeof(float));
	hipMalloc(&gpuptr_extforces, numnodes * 3 * sizeof(float));
	hipMalloc(&gpuptr_b, numnodes * 3 * sizeof(float));

	hipMalloc(&gpuptrR, numnodes * 3 * sizeof(float));
	hipMalloc(&gpuptrD, numnodes * 3 * sizeof(float));
	hipMalloc(&gpuptrQ, numnodes * 3 * sizeof(float));
	hipMalloc(&gpuptrVars, sizeof(CGVars));

	float coeffK = dt * BETA + dt * dt, coeffM = 1 + dt * ALPHA;
	float dt = 1.0/FPS;

	hipMemcpyToSymbol(HIP_SYMBOL("COEFFK"), &coeffK, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL("COEFFM"), &coeffM, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL("dt"), &dt, sizeof(float));


		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
			system("pause");
		}

	//testing
	//float F[3][3] = { {1,2,3}, {3,2,1}, {1,3,2} };
	//float R[3][3];
	//gpuComputePolarDecomposition(F,R);

	//for(int i=0;i<3;i++, printf("\n"))
	//	for(int j=0; j<3; j++)
	//		printf("%f ", R[i][j]);
	//system("pause");
}

__host__
void
gpuUploadVars(GPUElement* gpuElements, GPUNode* gpuNodes,float* xt, 
			  float* vt, float* extforces, int numnodes, int numelements)
{
	int numblocksperele = (numelements / BLOCK_SIZE) + 1;
	int numblockpernode = (numnodes / NODE_BLOCK_SIZE) + 1;

	hipMemcpy(gpuptrElements, gpuElements, numblocksperele * sizeof(GPUElement), hipMemcpyHostToDevice);
	hipMemcpy(gpuptrNodes, gpuNodes, numblockpernode * sizeof(GPUNode), hipMemcpyHostToDevice);
	hipMemcpy(gpuptr_xt, xt, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gpuptr_vt, vt, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gpuptr_extforces, extforces, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice);

			hipError_t error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
			system("pause");
		}
}

__host__
void
gpuDownloadVars(float* xt, int numnodes)
{
	hipMemcpy(xt, gpuptr_xt, numnodes * 3 * sizeof(float), hipMemcpyDeviceToHost);
}

__host__
void
gpuUploadExtForces(float* extforces, int numnodes)
{
	hipMemcpy(gpuptr_extforces, extforces, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice);

			hipError_t error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
			system("pause");
		}
}


__host__
void
gpuDestroyVars()
{
	hipFree(gpuptrElements);
	hipFree(gpuptrNodes);
	hipFree(gpuptrMulData);
	hipFree(gpuptr_xt);
	hipFree(gpuptr_vt);
	hipFree(gpuptr_extforces);
	hipFree(gpuptr_b);
	hipFree(gpuptrR);
	hipFree(gpuptrD);
	hipFree(gpuptrQ);
	hipFree(gpuptrVars);
}


__device__
void makeRK(float mat[12][12][BLOCK_SIZE], float R[3][3])
{
	int ltid = threadIdx.x;
	float RK[12][12];
	for(int i=0;i<4;i++)
	{
		for(int j=0;j<4;j++)
		{
			for(int a=0;a<3;a++)
			{
				for(int b=0;b<3;b++)
				{
					RK[a + i * 3][b + j * 3]= 0.0f;
						
					for(int c=0;c<3;c++)
						RK[a + i * 3][b + j * 3] += R[a][c] * mat[c + i * 3][b + j * 3][ltid];
				}
			}
		}
	}

	for(int i=0;i<12;i++)
		for(int j=0;j<12;j++)
			mat[i][j][ltid] = RK[i][j];
}


__device__
void makeRKRT(float mat[12][12][BLOCK_SIZE], float R[3][3])
{
	float RKRT[12][12];
	int ltid = threadIdx.x;

	for(int i=0;i<4;i++)
	{
		for(int j=i;j<4;j++)
		{
			for(int a=0;a<3;a++)
				for(int b=0;b<3;b++)
				{
					RKRT[a + i * 3][b + j * 3] = 0.0f;
					for(int c=0;c<3;c++)
						RKRT[a + i * 3][b + j * 3] += mat[a + i * 3][c + j * 3][ltid] * R[b][c]; //R(c, b) but its RT so, R(b , c)
				}
		}
	}
	//lower triangle
	for(int i=1;i<4;i++)
		for(int j=0;j<i;j++)
		{
			for(int a=0;a<3;a++)
				for(int b=0;b<3;b++)
						RKRT[a + i * 3][b + j * 3] = RKRT[b + j * 3][a + i * 3];
		}

	for(int i=0;i<12;i++)
		for(int j=0;j<12;j++)
			mat[i][j][ltid] = RKRT[i][j];
}


__device__
void mulSystem(GPUElement* elements, mulData* solverData, float* x)
{
	int bid = blockIdx.x;
	int ltid = threadIdx.x;

	GPUElement* t_ele = &(elements[bid]);
	mulData* t_solvedata = &(solverData[bid]);

	float nodes[12];

	for(int i=0;i<4;i++)
	{
		nodes[i * 3] = x[t_ele->nodeindex[i][ltid] * 3];
		nodes[i * 3 + 1] = x[t_ele->nodeindex[i][ltid] * 3 + 1];
		nodes[i * 3 + 2] = x[t_ele->nodeindex[i][ltid] * 3 + 2];
	}

	for(int i=0;i<12;i++)
	{
		t_solvedata->product[i][ltid] = 0;
		for(int j=0;j<12;j++)
			t_solvedata->product[i][ltid] += t_solvedata->system[i][j][ltid] * nodes[j];
	}
}

#define TETINDEX (node->elementindex[i][0][groupid] / BLOCK_SIZE)
#define TETINDEX2 (node->elementindex[i][0][groupid] % BLOCK_SIZE)
#define NODEINDEX (node->elementindex[i][1][groupid])

__device__
void mulSystemGather(GPUNode* nodes, mulData* solverData, float* x, int numnodes)
{
	int groupid = threadIdx.x / NODE_THREADS;
	int grouptid = threadIdx.x % NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_BLOCK_SIZE][4][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[groupid][grouptid];
	
	if(nodeno < numnodes)
	{

		cache[groupid][grouptid][0] = 0;
		cache[groupid][grouptid][1] = 0;
		cache[groupid][grouptid][2] = 0;

		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][groupid][grouptid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][groupid][grouptid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][groupid][grouptid];

			cache[groupid][grouptid][0] += solverData[tetindex].product[nodeindex * 3][tetindex2];
			cache[groupid][grouptid][1] += solverData[tetindex].product[nodeindex * 3 + 1][tetindex2];
			cache[groupid][grouptid][2] += solverData[tetindex].product[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{
			x[nodeno * 3]     = cache[groupid][0][0] + cache[groupid][1][0] + cache[groupid][2][0] + cache[groupid][3][0];
			x[nodeno * 3 + 1] = cache[groupid][0][1] + cache[groupid][1][1] + cache[groupid][2][1] + cache[groupid][3][1];
			x[nodeno * 3 + 2] = cache[groupid][0][2] + cache[groupid][1][2] + cache[groupid][2][2] + cache[groupid][3][2];
		}
	}

}

__device__
void dot(float*a, float*b, float* out, int n) 
{
	__shared__ float temp[DOT_BLOCK_SIZE];
	int index = threadIdx.x;
	int element = index;

	float tmp = 0;

	while(element < n)
	{
		tmp += a[element] * b[element];
		element += DOT_BLOCK_SIZE;
	}

	temp[index] = tmp;

	__syncthreads();


	int i = DOT_BLOCK_SIZE >> 1;
	while(i>0)
	{
		if(index < i)
			temp[index] += temp[index + i];
		__syncthreads();
		i>>=1;
	}

	if(index == 0)
		*out = temp[0];
}

//step 1
//precompute
__global__
void precompute(GPUElement* elements, mulData* solverData, float* xt, float* vt, float* extforces, int numelements)
{
	int tid = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	int bid = blockIdx.x;
	int ltid = threadIdx.x;

	if(tid < numelements)
	{
		GPUElement* t_ele = &(elements[bid]);
		mulData* t_solvedata = &(solverData[bid]);

		float nodalmass = t_ele->nodalmass[ltid];

		float nodes[12], vel[12], F[3][3]={0}, R[3][3] = {{1,0,0},{0,1,0},{0,0,1}};

		for(int i=0;i<4;i++)
		{
			nodes[i * 3] = xt[t_ele->nodeindex[i][ltid] * 3];
			nodes[i * 3 + 1] = xt[t_ele->nodeindex[i][ltid] * 3 + 1];
			nodes[i * 3 + 2] = xt[t_ele->nodeindex[i][ltid] * 3 + 2];
		}

		for(int i=0;i<4;i++)
		{
			vel[i * 3] = vt[t_ele->nodeindex[i][ltid] * 3];
			vel[i * 3 + 1] = vt[t_ele->nodeindex[i][ltid] * 3 + 1];
			vel[i * 3 + 2] = vt[t_ele->nodeindex[i][ltid] * 3 + 2];
		}

		for(int i=0;i<3;i++)
			for(int j=0;j<3;j++)
				for(int k=0;k<3;k++)
					F[i][j] += (nodes[k*3 + i] - nodes[9 + i]) * t_ele->undefShapeMatInv[k][j][ltid];

		gpuComputePolarDecomposition(F,R);


		for(int i=0;i<12;i++)
			for(int j=0;j<12;j++)
				t_solvedata->system[i][j][ltid] = t_ele->unwarpK[i][j][ltid];
	
		for(int i=0;i<4;i++)
		{
			t_solvedata->b[i * 3][ltid] = extforces[t_ele->nodeindex[i][ltid] * 3];
			t_solvedata->b[i * 3 + 1][ltid] = extforces[t_ele->nodeindex[i][ltid] * 3 + 1];
			t_solvedata->b[i * 3 + 2][ltid] = extforces[t_ele->nodeindex[i][ltid] * 3 + 2];
		}

		makeRK(t_solvedata->system, R);

		for(int i=0;i<12;i++)
			for(int j=0;j<12;j++)
				t_solvedata->b[i][ltid] += t_solvedata->system[i][j][ltid] * t_ele->x0[j][ltid];
	
		makeRKRT(t_solvedata->system, R);

		for(int i=0;i<12;i++)
			for(int j=0;j<12;j++)
				t_solvedata->b[i][ltid] -= t_solvedata->system[i][j][ltid] * nodes[j];

		for(int i=0;i<12;i++)
			t_solvedata->b[i][ltid] = t_solvedata->b[i][ltid] * dt + nodalmass * vel[i];

		//final system matrix
		for(int i=0;i<12;i++)
			for(int j=0;j<12;j++)
			{
				t_solvedata->system[i][j][ltid] *= COEFFK;
				if(i==j)
					t_solvedata->system[i][i][ltid] += COEFFM * nodalmass;
			}
	}
}

//step 2
//precompute
__global__
void gatherB(GPUNode* nodes, mulData* solverData, float* b, int numnodes)
{
	int groupid = threadIdx.x / NODE_THREADS;
	int grouptid = threadIdx.x % NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_BLOCK_SIZE][4][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[groupid][grouptid];
	
	if(nodeno < numnodes)
	{

		cache[groupid][grouptid][0] = 0;
		cache[groupid][grouptid][1] = 0;
		cache[groupid][grouptid][2] = 0;

		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][groupid][grouptid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][groupid][grouptid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][groupid][grouptid];

			cache[groupid][grouptid][0] += solverData[tetindex].b[nodeindex * 3][tetindex2];
			cache[groupid][grouptid][1] += solverData[tetindex].b[nodeindex * 3 + 1][tetindex2];
			cache[groupid][grouptid][2] += solverData[tetindex].b[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{
			b[nodeno * 3]     = cache[groupid][0][0] + cache[groupid][1][0] + cache[groupid][2][0] + cache[groupid][3][0];
			b[nodeno * 3 + 1] = cache[groupid][0][1] + cache[groupid][1][1] + cache[groupid][2][1] + cache[groupid][3][1];
			b[nodeno * 3 + 2] = cache[groupid][0][2] + cache[groupid][1][2] + cache[groupid][2][2] + cache[groupid][3][2];
		}
	}
}

//step 1
//init CG
// x = velocity
__global__
void
initAx(GPUElement* elements, mulData* solverData, float* x, int numelements)
{
	int tid = threadIdx.x + blockIdx.x * BLOCK_SIZE;

	if(tid < numelements)
	{
		mulSystem(elements, solverData, x);
	}
}

//step2
//init CG
__global__
void
initRandD(GPUNode* nodes, mulData* solverData, float* r, float* d, float* b, int numnodes)
{
	int groupid = threadIdx.x / NODE_THREADS;
	int grouptid = threadIdx.x % NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_BLOCK_SIZE][4][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[groupid][grouptid];
	
	if(nodeno < numnodes)
	{

		cache[groupid][grouptid][0] = 0;
		cache[groupid][grouptid][1] = 0;
		cache[groupid][grouptid][2] = 0;

		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][groupid][grouptid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][groupid][grouptid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][groupid][grouptid];

			cache[groupid][grouptid][0] += solverData[tetindex].product[nodeindex * 3][tetindex2];
			cache[groupid][grouptid][1] += solverData[tetindex].product[nodeindex * 3 + 1][tetindex2];
			cache[groupid][grouptid][2] += solverData[tetindex].product[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{
			r[nodeno * 3]     = cache[groupid][0][0] + cache[groupid][1][0] + cache[groupid][2][0] + cache[groupid][3][0];
			r[nodeno * 3 + 1] = cache[groupid][0][1] + cache[groupid][1][1] + cache[groupid][2][1] + cache[groupid][3][1];
			r[nodeno * 3 + 2] = cache[groupid][0][2] + cache[groupid][1][2] + cache[groupid][2][2] + cache[groupid][3][2];
		
			//r = b-Ax
			r[nodeno * 3] = b[nodeno * 3] - r[nodeno * 3];
			r[nodeno * 3 + 1] = b[nodeno * 3 + 1] - r[nodeno * 3 + 1];
			r[nodeno * 3 + 2] = b[nodeno * 3 + 2] - r[nodeno * 3 + 2];

			//d=r
			d[nodeno * 3] = r[nodeno * 3];
			d[nodeno * 3 + 1] = r[nodeno * 3 + 1];
			d[nodeno * 3 + 2] = r[nodeno * 3 + 2];
		}
	}

}

//step3
//init CG
//1 block, BLOCK_SIZE threads
__global__
void
initDeltaVars(CGVars* vars, float* r, int numnodes)
{
	__shared__ float rr;
	dot(r, r, &rr, numnodes * 3);
	
	if(threadIdx.x == 0)
	{
		vars->deltaNew = rr;
		vars->delta0 = vars->deltaNew;
	}
}

//step 4
//CG loop
//q = Ad
__global__
void
makeQprod(GPUElement* elements, mulData* solverData, float* d, int numelements)
{
	int tid = threadIdx.x + blockIdx.x * BLOCK_SIZE;

	if(tid < numelements)
	{
		mulSystem(elements, solverData, d);
	}
}

//step 5
//CG loop
//q = Ad
__global__
void
gatherQprod(GPUNode* nodes, mulData* solverData, float* q, int numnodes)
{
		mulSystemGather(nodes, solverData, q, numnodes);

}

//step 6
//CG Loop
//make vars
//1 block, BLOCK_SIZE threads
__global__
void
makeVars(CGVars* vars, float* d, float* q, float* r, int numnodes)
{
	float dq, rq, qq;
	dot(d,q,&dq,numnodes * 3);
	dot(r,q,&rq,numnodes * 3);
	dot(q,q,&qq,numnodes * 3);

	__syncthreads();

	if(threadIdx.x == 0)
	{
		vars->alpha = vars->deltaNew / dq;
		vars->deltaOld = vars->deltaNew;

		//r.r = r'.r' - 2*alpha*(r'.q) + alpha * alpha * (q.q)
		vars->deltaNew = vars->deltaNew - (2 * vars->alpha) * rq + (vars->alpha * vars->alpha) * qq;
		vars->beta = vars->deltaNew / vars->deltaOld;
	}
}

//step 7
//CG Loop
//make x, r, d
//x = velocity
__global__
void
makeXRandD(CGVars* vars, float *x, float* r, float* d, float* q, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x * VECTOR_BLOCK_SIZE;
	if(tid < numnodes)
	{
		float alpha = vars->alpha;
		float beta = vars->beta;

		x[tid * 3] = x[tid * 3] + alpha * d[tid * 3];
		x[tid * 3 + 1] = x[tid * 3 + 1] + alpha * d[tid * 3 + 1];
		x[tid * 3 + 2] = x[tid * 3 + 2] + alpha * d[tid * 3 + 2];

		r[tid * 3] = r[tid * 3] - alpha * q[tid * 3];
		r[tid * 3 + 1] = r[tid * 3 + 1] - alpha * q[tid * 3 + 1];
		r[tid * 3 + 2] = r[tid * 3 + 2] - alpha * q[tid * 3 + 2];

		d[tid * 3] = r[tid * 3] + beta * d[tid * 3];
		d[tid * 3 + 1] = r[tid * 3 + 1] + beta * d[tid * 3 + 1];
		d[tid * 3 + 2] = r[tid * 3 + 2] + beta * d[tid * 3 + 2];
	}
} 

//step 8
//make x(t+1)
__global__
void
integrate(float *x, float* v, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x * VECTOR_BLOCK_SIZE;
	if(tid < numnodes)
	{
		x[tid * 3] = x[tid * 3] + dt * v[tid * 3];
		x[tid * 3 + 1] = x[tid * 3 + 1] + dt * v[tid * 3 + 1];
		x[tid * 3 + 2] = x[tid * 3 + 2] + dt * v[tid * 3 + 2];
	}
}

__host__
void
gpuTimeStep(int numelements, int numnodes)
{
	const int num_blocks_ele = (numelements/BLOCK_SIZE) + 1;
	const int num_blocks_node = (numnodes/NODE_BLOCK_SIZE) + 1;
	const int num_blocks_vec = (numnodes/VECTOR_BLOCK_SIZE) + 1;

	hipError_t error;

	printf("Started\n");
	
	precompute<<<num_blocks_ele, BLOCK_SIZE>>>(gpuptrElements, gpuptrMulData, gpuptr_xt, gpuptr_vt, gpuptr_extforces, numelements);
	
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("1");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}


	gatherB<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptr_b, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("2");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	initAx<<<num_blocks_ele, BLOCK_SIZE>>>(gpuptrElements, gpuptrMulData, gpuptr_vt, numelements);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("3");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	initRandD<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptrR, gpuptrD, gpuptr_b, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("4");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	initDeltaVars<<<1, DOT_BLOCK_SIZE>>>(gpuptrVars, gpuptrR, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("5");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	int i=0;

	CGVars vars;
	hipMemcpy(&vars, gpuptrVars, sizeof(CGVars), hipMemcpyDeviceToHost);

	printf("Loop Started");

	while(i < MAX_ITER && vars.deltaNew > (EPSIL * EPSIL) * vars.delta0)
	{
		makeQprod<<<num_blocks_ele, BLOCK_SIZE>>>(gpuptrElements, gpuptrMulData, gpuptrD, numelements);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("6");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		gatherQprod<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptrQ, numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("7");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		makeVars<<<1, DOT_BLOCK_SIZE>>>(gpuptrVars, gpuptrD, gpuptrQ, gpuptrR, numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("8");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		makeXRandD<<<num_blocks_vec, VECTOR_BLOCK_SIZE>>>(gpuptrVars, gpuptr_vt, gpuptrR, gpuptrD, gpuptrQ, numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("9");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		hipMemcpy(&vars, gpuptrVars, sizeof(CGVars), hipMemcpyDeviceToHost);
		i++;

	}

	printf("Loop Ended: %d\n", i);

	integrate<<<num_blocks_vec, VECTOR_BLOCK_SIZE>>>(gpuptr_xt, gpuptr_vt, numnodes);
}

