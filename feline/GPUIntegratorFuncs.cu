#include "hip/hip_runtime.h"
#pragma once
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "defines.h"
#include "GPUDataStructs.cuh"
#include "GPUPolarDecompose.cu"

#define BLOCK_SIZE 256
#define ALPHA 0.1
#define BETA 0.1


GPUElement* gpuptrElements;
GPUNodes*   gpuptrNodes;
mulData*	gpuptrMulData;
float*   gpuptr_x0;//const
float*   gpuptr_xt;//dynamic
float*   gpuptr_vt;//dynamic
float*	 gpuptr_extforces;//dynamic


void
gpuInitVars(int numele, int numnodes)
{
	hipMalloc(&gpuptrElements, numele * sizeof(GPUElement));
	hipMalloc(&gpuptrNodes, numnodes * sizeof(GPUNodes));
	hipMalloc(&gpuptrMulData, numele * sizeof(mulData));
	hipMalloc(&gpuptr_xt, numnodes * 3 * sizeof(float));
	hipMalloc(&gpuptr_vt, numnodes * 3 * sizeof(float));
	hipMalloc(&gpuptr_extforces, numnodes * 3 * sizeof(float));
}

void
gpuUploadExtForces(float* extforces, int numnodes)
{
	hipMemcpy(gpuptr_extforces, extforces, numnodes*3*sizeof(float),hipMemcpyHostToDevice);
}

__device__
void makeRK(float mat[12][12], float R[3][3])
{
	float RK[12][12];
	for(int i=0;i<4;i++)
	{
		for(int j=0;j<4;j++)
		{
			for(int a=0;a<3;a++)
			{
				for(int b=0;b<3;b++)
				{
					RK[a + i * 3][b + j * 3]=0;
						
					for(int c=0;c<3;c++)
						RK[a + i * 3][b + j * 3] += R[a][c] * mat[c + i * 3][b + j * 3];
				}
			}
		}
	}

	for(int i=0;i<12;i++)
		for(int j=0;j<12;j++)
			mat[i][j] = RK[i][j];
}


__device__
void makeRKRT(float mat[12][12], float R[3][3])
{
	float RKRT[12][12];

	for(int i=0;i<4;i++)
	{
		for(int j=i;j<4;j++)
		{
			for(int a=0;a<3;a++)
				for(int b=0;b<3;b++)
				{
					RKRT[a + i * 3][b + j * 3] = 0;
					for(int c=0;c<3;c++)
						RKRT[a + i * 3][b + j * 3] += mat[a + i * 3][c + j * 3] * R[b][c]; //R(c, b) but its RT so, R(b , c)
				}
		}
	}
	//lower triangle
	for(int i=1;i<4;i++)
		for(int j=0;j<i;j++)
		{
			for(int a=0;a<3;a++)
				for(int b=0;b<3;b++)
						RKRT[a + i * 3][b + j * 3] = RKRT[b + j * 3][a + i * 3];
		}

	for(int i=0;i<12;i++)
		for(int j=0;j<12;j++)
			mat[i][j] = RKRT[i][j];
}

__device__
void makeA(float mat[12][12], float coeffK, float coeffM, float nodalMass)
{
	for(int i=0;i<12;i++)
		for(int j=0;j<12;j++)
		{
			mat[i][j] *= coeffK;
			if(i==j)
				mat[i][j] += coeffM * nodalMass;
		}
}


__global__
void precompute(GPUElement* elements, mulData* solverData, float* x0, float* xt, float* vt, float* extforces)
{
	int tid = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	GPUElement* t_ele = &(elements[tid]);
	mulData* t_solvedata = &(solverData[tid]);

	float nodes[12], vel[12], F[3][3], R[3][3];

	for(int i=0;i<4;i++)
	{
		nodes[i * 3] = xt[t_ele->nodeindex[i] * 3];
		nodes[i * 3 + 1] = xt[t_ele->nodeindex[i] * 3 + 1];
		nodes[i * 3 + 2] = xt[t_ele->nodeindex[i] * 3 + 2];
	}

	for(int i=0;i<4;i++)
	{
		vel[i * 3] = vt[t_ele->nodeindex[i] * 3];
		vel[i * 3 + 1] = vt[t_ele->nodeindex[i] * 3 + 1];
		vel[i * 3 + 2] = vt[t_ele->nodeindex[i] * 3 + 2];
	}

	for(int i=0;i<3;i++)
		for(int j=0;j<3;j++)
		{
			F[i][j] = nodes[j*3 + i] - nodes[9 + i];
		}

	gpuComputePolarDecomposition(F,R);


	for(int i=0;i<12;i++)
		for(int j=0;j<12;j++)
			t_solvedata->system[i][j] = t_ele->unwarpK[i][j];
	
	for(int i=0;i<4;i++)
	{
		t_ele->b[i * 3] = extforces[t_ele->nodeindex[i] * 3];
		t_ele->b[i * 3 + 1] = extforces[t_ele->nodeindex[i] * 3 + 1];
		t_ele->b[i * 3 + 2] = extforces[t_ele->nodeindex[i] * 3 + 2];
	}

	makeRK(t_solvedata->system, R);

	for(int i=0;i<12;i++)
		for(int j=0;j<12;j++)
			t_ele->b[i] += t_solvedata->system[i][j] * t_ele->x0[j];
	
	makeRKRT(t_solvedata->system, R);

	for(int i=0;i<12;i++)
		for(int j=0;j<12;j++)
			t_ele->b[i] -= t_solvedata->system[i][j] * nodes[j];

	for(int i=0;i<12;i++)
		t_ele->b[i] = t_ele->b[i] * DT + t_ele->nodalmass * vel[i];

	//todo: work in coeffK and coeffM
}

__global__
void collateB(GPUNode* nodes, GPUElement* elements, float* b)
{
	int tid = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	GPUNode* node = &(nodes[tid]);

	int n = node->n;

	for(int i=0;i<n;i++)
	{
		int tetindex = node->elementindex[i][0];
		int nodeindex = node->elementindex[i][1];

		b[tid * 3] = elements[tetindex].b[nodeindex * 3];
		b[tid * 3 + 1] = elements[tetindex].b[nodeindex * 3 + 1];
		b[tid * 3 + 2] = elements[tetindex].b[nodeindex * 3 + 2];
	}
}

void
timestep()
{
	
}

