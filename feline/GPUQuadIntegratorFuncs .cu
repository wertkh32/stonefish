#include "hip/hip_runtime.h"
#pragma once
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "defines.h"

#ifdef _QUAD_TET_

#include "GPUDataStructs.cuh"
#include "GPUPolarDecompose.cu"

//#define BLOCK_SIZE 512

#define ALPHA 0.4
#define BETA 0.5

#define MAX_ITER 20
#define EPSIL 0.05

__constant__ float COEFFK, COEFFM, dt;

GPUElement* gpuptrElements;
GPUNode*   gpuptrNodes;
mulData*	gpuptrMulData;
float*   gpuptr_xt;//dynamic
float*   gpuptr_vt;//dynamic
float*	 gpuptr_extforces;//dynamic
float*	 gpuptr_mass;//static
float*	 gpuptr_b;//dynamic
char*	 gpuptr_allowed;

//for CG
float* gpuptrR;
float* gpuptrD;
float* gpuptrQ;
CGVars* gpuptrVars;

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
		system("pause");
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__host__
void
gpuInitVars(int numele, int numnodes)
{
	int numblocksperele = (numele / BLOCK_SIZE) + 1;
	int numblockpernode = (numnodes / NODE_BLOCK_SIZE) + 1;

	//hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	HANDLE_ERROR( hipMalloc(&gpuptrElements, numblocksperele * sizeof(GPUElement)) );
	HANDLE_ERROR( hipMalloc(&gpuptrMulData, numblocksperele * sizeof(mulData)) );
	HANDLE_ERROR( hipMalloc(&gpuptrNodes, numblockpernode * sizeof(GPUNode)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_xt, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_vt, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_extforces, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_mass, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_b, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_allowed, numnodes * sizeof(char)) );


	HANDLE_ERROR( hipMalloc(&gpuptrR, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptrD, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptrQ, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptrVars, sizeof(CGVars)) );

	float ddt = 1.0/FPS;
	float coeffK = ddt * BETA + ddt * ddt, coeffM = 1 + ddt * ALPHA;
	

	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL("COEFFK"), &coeffK, sizeof(float)) );
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL("COEFFM"), &coeffM, sizeof(float)) );
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL("dt"), &ddt, sizeof(float)) );

		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
			system("pause");
		}
}

__host__
void
gpuUploadVars(GPUElement* gpuElements, GPUNode* gpuNodes,float* xt, 
			  float* vt, float* extforces, float* mass, char* allowed, int numnodes, int numelements)
{
	int numblocksperele = (numelements / BLOCK_SIZE) + 1;
	int numblockpernode = (numnodes / NODE_BLOCK_SIZE) + 1;

	HANDLE_ERROR( hipMemcpy(gpuptrElements, gpuElements, numblocksperele * sizeof(GPUElement), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptrNodes, gpuNodes, numblockpernode * sizeof(GPUNode), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_xt, xt, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_vt, vt, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_extforces, extforces, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_mass, mass, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_allowed, allowed, numnodes * sizeof(char), hipMemcpyHostToDevice) );

		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
			system("pause");
		}
}

__host__
void
gpuDownloadVars(float* xt, int numnodes)
{
	hipMemcpy(xt, gpuptr_xt, numnodes * 3 * sizeof(float), hipMemcpyDeviceToHost);
}

__host__
void
gpuUploadExtForces(float* extforces, int numnodes)
{
	hipMemcpy(gpuptr_extforces, extforces, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
		system("pause");
	}
}


__host__
void
gpuDestroyVars()
{
	hipFree(gpuptrElements);
	hipFree(gpuptrNodes);
	hipFree(gpuptrMulData);
	hipFree(gpuptr_xt);
	hipFree(gpuptr_vt);
	hipFree(gpuptr_extforces);
	hipFree(gpuptr_mass);
	hipFree(gpuptr_b);
	hipFree(gpuptrR);
	hipFree(gpuptrD);
	hipFree(gpuptrQ);
	hipFree(gpuptrVars);
}

__device__
void makeFU(float f0[30][BLOCK_SIZE], float R[3][3], float out[30])
{
	int ltid = threadIdx.x;
	float x[30];

	#pragma unroll 30
	for(int i=0;i<30;i++)
		x[i] = f0[i][ltid];

	#pragma unroll 10
	for(int i=0;i<10;i++)
		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			out[i*3 + j] = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
			out[i*3+j] += R[j][k] * x[i*3 + k];
		}		
}

//xt will be malnipulated and used as a temp array. do not further malnipulate xt
__device__
void makeRKRT(float system[30][30][BLOCK_SIZE], float R[3][3], float xt[30], float b[30])
{
	float temp[30];

	#pragma unroll 10
	for(int i=0;i<10;i++)
		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			temp[i*3 + j] = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
			temp[i*3+j] += R[k][j] * xt[i*3 + k]; //RT first
		}

	#pragma unroll 30
	for(int i=0;i<30;i++)
		xt[i] = 0;
	
	#pragma unroll 30
	for(int j=0;j<30;j++)
	{
		#pragma unroll 30
		for(int i=0;i<30;i++)
		xt[i] += system[j][i][threadIdx.x] * temp[j];
	}		

	#pragma unroll 10
	for(int i=0;i<10;i++)
		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			#pragma unroll 3
			for(int k=0;k<3;k++)
			b[i*3+j] -= R[j][k] * xt[i*3 + k];
		}

}

__device__
void mulSystem(GPUElement* elements, mulData* solverData, float* x, int numelements, int numnodes)
{
	int bid = blockIdx.x;
	int ltid = threadIdx.x % BLOCK_SIZE;
	int etid = threadIdx.x / BLOCK_SIZE;
	int tid = ltid + blockIdx.x * BLOCK_SIZE;
	
	GPUElement* t_ele = &(elements[bid]);
	mulData* t_solvedata = &(solverData[bid]);

	__shared__ float nodes[30][BLOCK_SIZE]; 
	__shared__ float R[3][3][BLOCK_SIZE];

	float temp[3];

	if(tid < numelements)
	{
		if(etid == 0)
		{
			#pragma unroll 3
			for(int i=0;i<3;i++)
				#pragma unroll 3
				for(int j=0;j<3;j++)
					R[i][j][ltid] = t_solvedata->R[i][j][ltid];
		}
	}

	__syncthreads();

	if(tid < numelements)
	{

		//first batch
		//rotate by x by RT first
		int index = t_ele->nodeindex[etid][ltid];

		float temp2[3];
		temp2[0] = x[index];
		temp2[1] = x[index + numnodes];
		temp2[2] = x[index + numnodes * 2];

		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			temp[j] = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
			temp[j] += R[k][j][ltid] * temp2[k];
		}

		nodes[etid * 3][ltid] = temp[0];
		nodes[etid * 3 + 1][ltid] = temp[1];
		nodes[etid * 3 + 2][ltid] = temp[2];

		//START OF SECOND BATCH//////////////////////////
		index = t_ele->nodeindex[etid+THREADS_PER_ELE][ltid];

		temp2[0] = x[index];
		temp2[1] = x[index + numnodes];
		temp2[2] = x[index + numnodes * 2];

		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			temp[j] = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
			temp[j] += R[k][j][ltid] * temp2[k];
		}

		nodes[(etid+THREADS_PER_ELE) * 3][ltid] = temp[0];
		nodes[(etid+THREADS_PER_ELE) * 3 + 1][ltid] = temp[1];
		nodes[(etid+THREADS_PER_ELE) * 3 + 2][ltid] = temp[2];

		////////////////////////////////////////////////////////////		
	}

	__syncthreads();

	if(tid < numelements)
	{

		///FIRST BATCH///////////////////////////////
		temp[0] = 0;
		temp[1] = 0;	
		temp[2] = 0;
		
		#pragma unroll 6
		for(int j=0;j<30;j++)
		{
			#pragma unroll 3
			for(int i=0;i<3;i++)
				temp[i] += t_ele->system[j][etid * 3 + i][ltid] * nodes[j][ltid];
		}

		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			float temp3 = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
				temp3 += R[j][k][ltid] * temp[k];

			t_solvedata->product[etid*3 + j][ltid] = temp3 * COEFFK;
		}

		//SECOND BATCH///////////////////////////////////////////////
		temp[0] = 0;
		temp[1] = 0;	
		temp[2] = 0;
		
		#pragma unroll 6
		for(int j=0;j<30;j++)
		{
			#pragma unroll 3
			for(int i=0;i<3;i++)
				temp[i] += t_ele->system[j][(etid+THREADS_PER_ELE) * 3 + i][ltid] * nodes[j][ltid];
		}

		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			float temp3 = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
				temp3 += R[j][k][ltid] * temp[k];

			t_solvedata->product[(etid+THREADS_PER_ELE)*3 + j][ltid] = temp3 * COEFFK;
		}


	}
}

__device__
void dot(float*a, float*b, float* out, int n) 
{
	__shared__ float temp[DOT_BLOCK_SIZE];
	int index = threadIdx.x;
	int element = index;

	float tmp = 0;

	while(element < n)
	{
		tmp += a[element] * b[element];
		element += DOT_BLOCK_SIZE;
	}

	temp[index] = tmp;

	__syncthreads();


	int i = DOT_BLOCK_SIZE >> 1;
	while(i>0)
	{
		if(index < i)
			temp[index] += temp[index + i];
		__syncthreads();
		i>>=1;
	}

	if(index == 0)
		*out = temp[0];
}

//step 1
//precompute
__global__
void precompute(GPUElement* elements, mulData* solverData, float* xt, int numelements, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	int bid = blockIdx.x;
	int ltid = threadIdx.x;

	if(tid < numelements)
	{
		GPUElement* t_ele = &(elements[bid]);
		mulData* t_solvedata = &(solverData[bid]);

		float nodes[30], b[30], R[3][3]={0}, D[3][3];
			

		#pragma unroll 10
		for(int i=0;i<10;i++)
		{
			int index = t_ele->nodeindex[i][ltid];
			nodes[i * 3] = xt[index];
			nodes[i * 3 + 1] = xt[index + numnodes];
			nodes[i * 3 + 2] = xt[index + numnodes * 2];
		}

		#pragma unroll 3
		for(int i=0;i<3;i++)
			#pragma unroll 3
			for(int j=0;j<3;j++)
				D[i][j] = t_ele->B[i][j][ltid];

		#pragma unroll 3
		for(int i=0;i<3;i++)
			#pragma unroll 3
			for(int j=0;j<3;j++)
					R[i][j] = (nodes[i] - nodes[9 + i]) * D[0][j] + 
							  (nodes[3 + i] - nodes[9 + i]) * D[1][j] + 
							  (nodes[6 + i] - nodes[9 + i]) * D[2][j];

		gpuComputePolarDecomposition(R);
	
		#pragma unroll 3
		for(int i=0;i<3;i++)
			#pragma unroll 3
			for(int j=0;j<3;j++)
				t_solvedata->R[i][j][ltid] =  R[i][j];

		makeFU(t_ele->f0,R,b);
	
		makeRKRT(t_ele->system, R, nodes, b);

		#pragma unroll 30
		for(int i=0;i<30;i++)
			t_solvedata->b[i][ltid] = b[i] * dt;

	}
}

//step 2
//precompute
__global__
void gatherB(GPUNode* nodes, mulData* solverData, float* b, float* mass, float* vt, float* extforces, char* allowed,int numnodes)
{
	int groupid = threadIdx.x % NODE_BLOCK_SIZE;// / NODE_THREADS;
	int grouptid = threadIdx.x / NODE_BLOCK_SIZE; //% NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_THREADS][NODE_BLOCK_SIZE][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[grouptid][groupid];
	
	if(nodeno < numnodes)
	{

		cache[grouptid][groupid][0] = 0;
		cache[grouptid][groupid][1] = 0;
		cache[grouptid][groupid][2] = 0;


		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][grouptid][groupid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][grouptid][groupid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][grouptid][groupid];

			cache[grouptid][groupid][0] += solverData[tetindex].b[nodeindex * 3][tetindex2];
			cache[grouptid][groupid][1] += solverData[tetindex].b[nodeindex * 3 + 1][tetindex2];
			cache[grouptid][groupid][2] += solverData[tetindex].b[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{
			b[nodeno]     = cache[0][groupid][0] + cache[1][groupid][0] + mass[nodeno] * vt[nodeno] + extforces[nodeno] * dt;
			b[nodeno + numnodes] = cache[0][groupid][1] + cache[1][groupid][1] + mass[nodeno + numnodes] * vt[nodeno + numnodes] + extforces[nodeno + numnodes] * dt;
			b[nodeno + numnodes * 2] = cache[0][groupid][2] + cache[1][groupid][2] + mass[nodeno + numnodes * 2] * vt[nodeno + numnodes * 2] + extforces[nodeno + numnodes * 2] * dt;

			char bitsy = allowed[nodeno];
			if(bitsy & 1)
				vt[nodeno] = 0;
			if(bitsy & 2)
				vt[nodeno + numnodes] = 0;
			if(bitsy & 4)
				vt[nodeno + numnodes * 2] = 0;

		}
	}
}

//step 1
//init CG
// x = velocity
__global__
void
initAx(GPUElement* elements, mulData* solverData, float* x, int numelements, int numnodes)
{
		mulSystem(elements, solverData, x, numelements, numnodes);
}

//step2
//init CG
__global__
void
initRandD(GPUNode* nodes, mulData* solverData, float* r, float* d, float* b, float* mass, float* vt, char* allowed, int numnodes)
{
	int groupid = threadIdx.x % NODE_BLOCK_SIZE;// / NODE_THREADS;
	int grouptid = threadIdx.x / NODE_BLOCK_SIZE; //% NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_THREADS][NODE_BLOCK_SIZE][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[grouptid][groupid];
	
	if(nodeno < numnodes)
	{

		cache[grouptid][groupid][0] = 0;
		cache[grouptid][groupid][1] = 0;
		cache[grouptid][groupid][2] = 0;

		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][grouptid][groupid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][grouptid][groupid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][grouptid][groupid];

			cache[grouptid][groupid][0] += solverData[tetindex].product[nodeindex * 3][tetindex2];
			cache[grouptid][groupid][1] += solverData[tetindex].product[nodeindex * 3 + 1][tetindex2];
			cache[grouptid][groupid][2] += solverData[tetindex].product[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{	
			char bitsy = allowed[nodeno];

			//r = b-Ax
			float r0 =  (bitsy & 1) ? 0 : (b[nodeno] - (cache[0][groupid][0] + cache[1][groupid][0] + mass[nodeno] * vt[nodeno] * COEFFM));
			float r1 =  (bitsy & 2) ? 0 : (b[nodeno + numnodes] - (cache[0][groupid][1] + cache[1][groupid][1] + mass[nodeno + numnodes] * vt[nodeno + numnodes] * COEFFM));
			float r2 =  (bitsy & 4) ? 0 : (b[nodeno + numnodes * 2] - (cache[0][groupid][2] + cache[1][groupid][2] + mass[nodeno + numnodes * 2] * vt[nodeno + numnodes * 2] * COEFFM));

			r[nodeno] = r0;
			r[nodeno + numnodes] = r1;
			r[nodeno + numnodes * 2] = r2;

			//d=r
			d[nodeno] = r0;
			d[nodeno + numnodes] = r1;
			d[nodeno + numnodes * 2] = r2;
		}
	}

}

//step3
//init CG
//1 block, BLOCK_SIZE threads
__global__
void
initDeltaVars(CGVars* vars, float* r, int numnodes)
{
	__shared__ float rr;
	dot(r, r, &rr, numnodes * 3);
	
	if(threadIdx.x == 0)
	{
		vars->deltaNew = rr;
		vars->delta0 = vars->deltaNew;
	}
}

//step 4
//CG loop
//q = Ad
__global__
void
makeQprod(GPUElement* elements, mulData* solverData, float* d, int numelements, int numnodes)
{
		mulSystem(elements, solverData, d, numelements, numnodes);
}

//step 5
//CG loop
//q = Ad
__global__
void
gatherQprod(GPUNode* nodes, mulData* solverData, float* q, float* mass, float* d, char* allowed, int numnodes)
{
	int groupid = threadIdx.x % NODE_BLOCK_SIZE;// / NODE_THREADS;
	int grouptid = threadIdx.x / NODE_BLOCK_SIZE; //% NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_THREADS][NODE_BLOCK_SIZE][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[grouptid][groupid];
	
	if(nodeno < numnodes)
	{

		cache[grouptid][groupid][0] = 0;
		cache[grouptid][groupid][1] = 0;
		cache[grouptid][groupid][2] = 0;

		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][grouptid][groupid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][grouptid][groupid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][grouptid][groupid];

			cache[grouptid][groupid][0] += solverData[tetindex].product[nodeindex * 3][tetindex2];
			cache[grouptid][groupid][1] += solverData[tetindex].product[nodeindex * 3 + 1][tetindex2];
			cache[grouptid][groupid][2] += solverData[tetindex].product[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{
			char bitsy = allowed[nodeno];
			q[nodeno]     = (bitsy & 1) ? 0 : (cache[0][groupid][0] + cache[1][groupid][0] + mass[nodeno] * d[nodeno] * COEFFM);
			q[nodeno + numnodes] = (bitsy & 2) ? 0 : (cache[0][groupid][1] + cache[1][groupid][1] + mass[nodeno + numnodes] * d[nodeno + numnodes] * COEFFM);
			q[nodeno + numnodes * 2] = (bitsy & 4) ? 0 : (cache[0][groupid][2] + cache[1][groupid][2] + mass[nodeno + numnodes * 2] * d[nodeno + numnodes * 2] * COEFFM);
		}
	}

}

//step 6
//CG Loop
//make vars
//1 block, BLOCK_SIZE threads
__global__
void
makeVars(CGVars* vars, float* d, float* q, float* r, int numnodes)
{
	float dq, rq, qq;
	dot(d,q,&dq,numnodes * 3);
	dot(r,q,&rq,numnodes * 3);
	dot(q,q,&qq,numnodes * 3);

	__syncthreads();

	if(threadIdx.x == 0)
	{
		vars->alpha = vars->deltaNew / dq;
		vars->deltaOld = vars->deltaNew;

		//r.r = r'.r' - 2*alpha*(r'.q) + alpha * alpha * (q.q)
		vars->deltaNew = vars->deltaNew - (2 * vars->alpha) * rq + (vars->alpha * vars->alpha) * qq;
		vars->beta = vars->deltaNew / vars->deltaOld;
	}
}

//step 7
//CG Loop
//make x, r, d
//x = velocity
__global__
void
makeXRandD(CGVars* vars, float *x, float* r, float* d, float* q, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x * VECTOR_BLOCK_SIZE;
	if(tid < numnodes)
	{
		float alpha = vars->alpha;
		float beta = vars->beta;
		float d1,d2,d3;
		float r1,r2,r3;

		d1 = d[tid];
		d2 =  d[tid + numnodes];
		d3 = d[tid + numnodes * 2];

		x[tid] = x[tid] + alpha * d1;
		x[tid + numnodes] = x[tid + numnodes] + alpha * d2;
		x[tid + numnodes * 2] = x[tid + numnodes * 2] + alpha * d3;

		r1 = r[tid] - alpha * q[tid];
		r2 = r[tid + numnodes] - alpha * q[tid + numnodes];
		r3 = r[tid + numnodes * 2] - alpha * q[tid + numnodes * 2];

		d[tid] = r1 + beta * d1;
		d[tid + numnodes] = r2 + beta * d2;
		d[tid + numnodes * 2] = r3 + beta * d3;

		r[tid] = r1;
		r[tid + numnodes] = r2;
		r[tid + numnodes * 2] = r3;
	}
} 

//step 8
//make x(t+1)
__global__
void
integrate(float *x, float* v, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x * VECTOR_BLOCK_SIZE;
	if(tid < numnodes)
	{
		x[tid] = x[tid] + dt * v[tid];
		x[tid + numnodes] = x[tid + numnodes] + dt * v[tid + numnodes];
		x[tid + numnodes * 2] = x[tid + numnodes * 2] + dt * v[tid + numnodes * 2];
	}
}

__host__
void
gpuTimeStep(int numelements, int numnodes)
{
	const int num_blocks_ele = (numelements/BLOCK_SIZE) + 1;
	const int num_blocks_node = (numnodes/NODE_BLOCK_SIZE) + 1;
	const int num_blocks_vec = (numnodes/VECTOR_BLOCK_SIZE) + 1;

	hipError_t error;

	printf("Started\n");
	
	precompute<<<num_blocks_ele, BLOCK_SIZE>>>(gpuptrElements, gpuptrMulData, gpuptr_xt, numelements, numnodes);
	
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("1");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}


	gatherB<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptr_b, gpuptr_mass, gpuptr_vt, gpuptr_extforces, gpuptr_allowed, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("2");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	initAx<<<num_blocks_ele, THREADS_PER_BLOCK>>>(gpuptrElements, gpuptrMulData, gpuptr_vt, numelements, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("3");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	initRandD<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptrR, gpuptrD, gpuptr_b, gpuptr_mass, gpuptr_vt,  gpuptr_allowed,numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("4");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	initDeltaVars<<<1, DOT_BLOCK_SIZE>>>(gpuptrVars, gpuptrR, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("5");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	int i=0;

	CGVars vars;
	hipMemcpy(&vars, gpuptrVars, sizeof(CGVars), hipMemcpyDeviceToHost);

	printf("Loop Started");
	
	while(i < MAX_ITER && vars.deltaNew > (EPSIL * EPSIL) * vars.delta0)
	{
		makeQprod<<<num_blocks_ele, THREADS_PER_BLOCK>>>(gpuptrElements, gpuptrMulData, gpuptrD, numelements, numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("6");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		gatherQprod<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptrQ, gpuptr_mass, gpuptrD, gpuptr_allowed,numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("7");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		makeVars<<<1, DOT_BLOCK_SIZE>>>(gpuptrVars, gpuptrD, gpuptrQ, gpuptrR, numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("8");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		makeXRandD<<<num_blocks_vec, VECTOR_BLOCK_SIZE>>>(gpuptrVars, gpuptr_vt, gpuptrR, gpuptrD, gpuptrQ, numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("9");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		hipMemcpy(&vars, gpuptrVars, sizeof(CGVars), hipMemcpyDeviceToHost);
		i++;

	}
	
	printf("Loop Ended: %d\n", i);

	integrate<<<num_blocks_vec, VECTOR_BLOCK_SIZE>>>(gpuptr_xt, gpuptr_vt, numnodes);
}


#endif