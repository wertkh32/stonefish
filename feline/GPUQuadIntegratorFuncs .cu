#include "hip/hip_runtime.h"
#pragma once
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "defines.h"

#ifdef _QUAD_TET_

#include "GPUDataStructs.cuh"
#include "GPUPolarDecompose.cu"

//#define BLOCK_SIZE 512

#define ZERO_EPS 0.000001

#define ALPHA 0.01
#define BETA 0.01

#define MAX_ITER 20
#define EPSIL 0.5

__constant__ float COEFFK, COEFFM, dt, S[16];

GPUElement* gpuptrElements;
GPUNode*   gpuptrNodes;
mulData*	gpuptrMulData;
float*   gpuptr_xt;//dynamic
float*   gpuptr_vt;//dynamic
float*	 gpuptr_extforces;//dynamic
float*	 gpuptr_mass;//static
float*	 gpuptr_b;//dynamic
char*	 gpuptr_allowed;

float*	gpuptr_minv;

//for CG
float* gpuptrR;
float* gpuptrD;
float* gpuptrQ;
CGVars* gpuptrVars;

//for debug
float* debugbuf;

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
		system("pause");
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__host__
void
gpuInitVars(int numele, int numnodes)
{
	int numblocksperele = (numele / BLOCK_SIZE) + 1;
	int numblockpernode = (numnodes / NODE_BLOCK_SIZE) + 1;

	//hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	HANDLE_ERROR( hipMalloc(&gpuptrElements, numblocksperele * sizeof(GPUElement)) );
	HANDLE_ERROR( hipMalloc(&gpuptrMulData, numblocksperele * sizeof(mulData)) );
	HANDLE_ERROR( hipMalloc(&gpuptrNodes, numblockpernode * sizeof(GPUNode)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_xt, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_vt, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_extforces, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_mass, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_b, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_allowed, numnodes * sizeof(char)) );
	HANDLE_ERROR( hipMalloc(&gpuptr_minv, numnodes * 3 * sizeof(float)) );

	HANDLE_ERROR( hipMalloc(&gpuptrR, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptrD, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptrQ, numnodes * 3 * sizeof(float)) );
	HANDLE_ERROR( hipMalloc(&gpuptrVars, sizeof(CGVars)) );

	debugbuf = (float*)malloc(numnodes * 3 * sizeof(float));


	float ddt = 1.0/FPS;
	float coeffK = ddt * BETA + ddt * ddt, coeffM = 1 + ddt * ALPHA;
	
	float a = ((5.0 + 3.0 * sqrt(5.0))/20.) * 4.0;
	float b = ((5.0 - sqrt(5.0))/20.) * 4.0;

	float _S[16] = { a,b,b,b,
					  b,a,b,b,
					  b,b,a,b,
					  b,b,b,a };

	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL("COEFFK"), &coeffK, sizeof(float)) );
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL("COEFFM"), &coeffM, sizeof(float)) );
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL("dt"), &ddt, sizeof(float)) );
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL("S"), _S, sizeof(float) * 16) );

		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
			system("pause");
		}
}

__host__
void
gpuUploadVars(GPUElement* gpuElements, GPUNode* gpuNodes,float* xt, 
			  float* vt, float* extforces, float* mass, char* allowed, int numnodes, int numelements)
{
	int numblocksperele = (numelements / BLOCK_SIZE) + 1;
	int numblockpernode = (numnodes / NODE_BLOCK_SIZE) + 1;

	HANDLE_ERROR( hipMemcpy(gpuptrElements, gpuElements, numblocksperele * sizeof(GPUElement), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptrNodes, gpuNodes, numblockpernode * sizeof(GPUNode), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_xt, xt, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_vt, vt, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_extforces, extforces, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_mass, mass, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(gpuptr_allowed, allowed, numnodes * sizeof(char), hipMemcpyHostToDevice) );

		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
			system("pause");
		}
}

__host__
void inspectGPUBuffer(float* gpubuf,int numnodes)
{
	hipMemcpy(debugbuf, gpubuf, numnodes * 3 * sizeof(float), hipMemcpyDeviceToHost);

	for(int i=0;i<numnodes * 3;i++)
		printf("%f ",debugbuf[i]);
	printf("\n");
	system("pause");
}

__host__
void
gpuDownloadVars(float* xt, int numnodes)
{
	hipMemcpy(xt, gpuptr_xt, numnodes * 3 * sizeof(float), hipMemcpyDeviceToHost);
}

__host__
void
gpuUploadExtForces(float* extforces, int numnodes)
{
	hipMemcpy(gpuptr_extforces, extforces, numnodes * 3 * sizeof(float), hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
		system("pause");
	}
}


__host__
void
gpuDestroyVars()
{
	hipFree(gpuptrElements);
	hipFree(gpuptrNodes);
	hipFree(gpuptrMulData);
	hipFree(gpuptr_xt);
	hipFree(gpuptr_vt);
	hipFree(gpuptr_extforces);
	hipFree(gpuptr_mass);
	hipFree(gpuptr_b);
	hipFree(gpuptrR);
	hipFree(gpuptrD);
	hipFree(gpuptrQ);
	hipFree(gpuptrVars);
}

#ifdef _BERSTEIN_POLY_
__device__
void mulSystem(GPUElement* elements, mulData* solverData, float* x, int numelements, int numnodes)
{
	int bid = blockIdx.x;
	int ltid = threadIdx.x % BLOCK_SIZE;
	int etid = threadIdx.x / BLOCK_SIZE;
	int tid = ltid + blockIdx.x * BLOCK_SIZE;
	
	GPUElement* t_ele = &(elements[bid]);
	mulData* t_solvedata = &(solverData[bid]);

	__shared__ float nodes[30][BLOCK_SIZE]; 
	__shared__ float R[3][3][BLOCK_SIZE];

	float temp[3];

	if(tid < numelements)
	{
		if(etid == 0)
		{
			#pragma unroll 3
			for(int i=0;i<3;i++)
				#pragma unroll 3
				for(int j=0;j<3;j++)
					R[i][j][ltid] = t_solvedata->R[i][j][ltid];
		}
	}

	__syncthreads();

	if(tid < numelements)
	{

		//first batch
		//rotate by x by RT first
		int index = t_ele->nodeindex[etid][ltid];

		float temp2[3];
		temp2[0] = x[index];
		temp2[1] = x[index + numnodes];
		temp2[2] = x[index + numnodes * 2];

		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			temp[j] = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
			temp[j] += R[k][j][ltid] * temp2[k];
		}

		nodes[etid * 3][ltid] = temp[0];
		nodes[etid * 3 + 1][ltid] = temp[1];
		nodes[etid * 3 + 2][ltid] = temp[2];

		//START OF SECOND BATCH//////////////////////////
		index = t_ele->nodeindex[etid+THREADS_PER_ELE][ltid];

		temp2[0] = x[index];
		temp2[1] = x[index + numnodes];
		temp2[2] = x[index + numnodes * 2];

		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			temp[j] = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
			temp[j] += R[k][j][ltid] * temp2[k];
		}

		nodes[(etid+THREADS_PER_ELE) * 3][ltid] = temp[0];
		nodes[(etid+THREADS_PER_ELE) * 3 + 1][ltid] = temp[1];
		nodes[(etid+THREADS_PER_ELE) * 3 + 2][ltid] = temp[2];

		////////////////////////////////////////////////////////////		
	}

	__syncthreads();

	if(tid < numelements)
	{

		///FIRST BATCH///////////////////////////////
		temp[0] = 0;
		temp[1] = 0;	
		temp[2] = 0;
		//symmetric so exploit
		#pragma unroll 6
		for(int j=0;j<30;j++)
		{
			#pragma unroll 3
			for(int i=0;i<3;i++)
				temp[i] += t_ele->system[j][etid * 3 + i][ltid] * nodes[j][ltid];
		}

		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			float temp3 = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
				temp3 += R[j][k][ltid] * temp[k];

			t_solvedata->product[etid*3 + j][ltid] = temp3;
		}

		//SECOND BATCH///////////////////////////////////////////////
		temp[0] = 0;
		temp[1] = 0;	
		temp[2] = 0;
		
		#pragma unroll 6
		for(int j=0;j<30;j++)
		{
			#pragma unroll 3
			for(int i=0;i<3;i++)
				temp[i] += t_ele->system[j][(etid+THREADS_PER_ELE) * 3 + i][ltid] * nodes[j][ltid];
		}

		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			float temp3 = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
				temp3 += R[j][k][ltid] * temp[k];

			t_solvedata->product[(etid+THREADS_PER_ELE)*3 + j][ltid] = temp3;
		}


	}
}
#endif



#ifdef _GAUSSIAN_QUADRATURE_

__device__
void mulK(float x[30], float b[4][3][BLOCK_SIZE], float c1[BLOCK_SIZE], float c2[BLOCK_SIZE])
{
	int ltid = threadIdx.x % BLOCK_SIZE;
	int etid = threadIdx.x / BLOCK_SIZE;
	float temp[6];
	float temp2[6];

	float con1 = c1[ltid];
	float con2 = c2[ltid];
	float con3 = (con1 - con2)/2.0;

	float* s = &S[etid * 4];

	float dndx[3][10];
					
	dndx[0][0] = (s[0]-1.0) * b[0][0][ltid];
	dndx[0][1] = (s[1]-1.0) * b[1][0][ltid];
	dndx[0][2] = (s[2]-1.0) * b[2][0][ltid];
	dndx[0][3] = (s[3]-1.0) * b[3][0][ltid];
	dndx[0][4] = (b[1][0][ltid] * s[0] +  b[0][0][ltid] * s[1]);
	dndx[0][5] = (b[2][0][ltid] * s[1] +  b[1][0][ltid] * s[2]);
	dndx[0][6] = (b[0][0][ltid] * s[2] +  b[2][0][ltid] * s[0]);
	dndx[0][7] = (b[3][0][ltid] * s[0] +  b[0][0][ltid] * s[3]);
	dndx[0][8] = (b[3][0][ltid] * s[1] +  b[1][0][ltid] * s[3]);
	dndx[0][9] = (b[3][0][ltid] * s[2] +  b[2][0][ltid] * s[3]);
	
	dndx[1][0] = (s[0]-1.0) * b[0][1][ltid];
	dndx[1][1] = (s[1]-1.0) * b[1][1][ltid];
	dndx[1][2] = (s[2]-1.0) * b[2][1][ltid];
	dndx[1][3] = (s[3]-1.0) * b[3][1][ltid];
	dndx[1][4] = (b[1][1][ltid] * s[0] +  b[0][1][ltid] * s[1]);
	dndx[1][5] = (b[2][1][ltid] * s[1] +  b[1][1][ltid] * s[2]);
	dndx[1][6] = (b[0][1][ltid] * s[2] +  b[2][1][ltid] * s[0]);
	dndx[1][7] = (b[3][1][ltid] * s[0] +  b[0][1][ltid] * s[3]);
	dndx[1][8] = (b[3][1][ltid] * s[1] +  b[1][1][ltid] * s[3]);
	dndx[1][9] = (b[3][1][ltid] * s[2] +  b[2][1][ltid] * s[3]);
	
	dndx[2][0] = (s[0]-1.0) * b[0][2][ltid];
	dndx[2][1] = (s[1]-1.0) * b[1][2][ltid];
	dndx[2][2] = (s[2]-1.0) * b[2][2][ltid];
	dndx[2][3] = (s[3]-1.0) * b[3][2][ltid];
	dndx[2][4] = (b[1][2][ltid] * s[0] +  b[0][2][ltid] * s[1]);
	dndx[2][5] = (b[2][2][ltid] * s[1] +  b[1][2][ltid] * s[2]);
	dndx[2][6] = (b[0][2][ltid] * s[2] +  b[2][2][ltid] * s[0]);
	dndx[2][7] = (b[3][2][ltid] * s[0] +  b[0][2][ltid] * s[3]);
	dndx[2][8] = (b[3][2][ltid] * s[1] +  b[1][2][ltid] * s[3]);
	dndx[2][9] = (b[3][2][ltid] * s[2] +  b[2][2][ltid] * s[3]);					   


	temp[0] = 0;
	temp[1] = 0;
	temp[2] = 0;

	
	#pragma unroll 10
	for(int j=0;j<10;j++)
		#pragma unroll 3
		for(int i=0;i<3;i++)
		temp[i] += dndx[i][j] * x[j * 3 + i];

	temp[3] = 0;
	temp[4] = 0;
	temp[5] = 0;

	#pragma unroll 10
	for(int j=0;j<10;j++)
	{
		temp[3] += dndx[1][j] * x[j * 3] + dndx[0][j] * x[j * 3 + 1];
		temp[4] += dndx[2][j] * x[j * 3 + 1] + dndx[1][j] * x[j * 3 + 2];
		temp[5] += dndx[2][j] * x[j * 3] + dndx[0][j] * x[j * 3 + 2];
	}

	temp2[0] = temp[0] * con1 + temp[1] * con2 + temp[2] * con2;
	temp2[1] = temp[0] * con2 + temp[1] * con1 + temp[2] * con2;
	temp2[2] = temp[0] * con2 + temp[1] * con2 + temp[2] * con1;
	temp2[3] = temp[3] * con3;
	temp2[4] = temp[4] * con3;
	temp2[5] = temp[5] * con3;

	#pragma unroll 10
	for(int i=0;i<10;i++)
	{
		x[i * 3] =     dndx[0][i] * temp2[0] + dndx[1][i] * temp2[3] + dndx[2][i] * temp2[5];
		x[i * 3 + 1] = dndx[1][i] * temp2[1] + dndx[0][i] * temp2[3] + dndx[2][i] * temp2[4];
		x[i * 3 + 2] = dndx[2][i] * temp2[2] + dndx[1][i] * temp2[4] + dndx[0][i] * temp2[5];
	}

}

__device__
void mulSystem(GPUElement* elements, mulData* solverData, float* x, int numelements, int numnodes)
{
	int bid = blockIdx.x;
	int ltid = threadIdx.x % BLOCK_SIZE;
	int etid = threadIdx.x / BLOCK_SIZE;
	int tid = ltid + blockIdx.x * BLOCK_SIZE;
	
	GPUElement* t_ele = &(elements[bid]);
	mulData* t_solvedata = &(solverData[bid]);

	__shared__ float nodes[30][BLOCK_SIZE]; 
	__shared__ float R[3][3][BLOCK_SIZE];
	__shared__ float B[4][3][BLOCK_SIZE];

	float temp[3];
	float out[30];

	if(tid < numelements)
	{
		if(etid == 0)
		{
			#pragma unroll 3
			for(int i=0;i<3;i++)
				#pragma unroll 3
				for(int j=0;j<3;j++)
					R[i][j][ltid] = t_solvedata->R[i][j][ltid];

			#pragma unroll 3
			for(int i=0;i<3;i++)
				#pragma unroll 3
				for(int j=0;j<3;j++)
					B[i][j][ltid] = t_ele->B[i][j][ltid];

			B[3][0][ltid] = -B[0][0][ltid]-B[1][0][ltid]-B[2][0][ltid];
			B[3][1][ltid] = -B[0][1][ltid]-B[1][1][ltid]-B[2][1][ltid];
			B[3][2][ltid] = -B[0][2][ltid]-B[1][2][ltid]-B[2][2][ltid];
		}
	}

	__syncthreads();

	if(tid < numelements)
	{

		//first batch
		//rotate by x by RT first
		int index = t_ele->nodeindex[etid][ltid];

		float temp2[3];
		temp2[0] = x[index];
		temp2[1] = x[index + numnodes];
		temp2[2] = x[index + numnodes * 2];

		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			temp[j] = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
			temp[j] += R[k][j][ltid] * temp2[k];
		}

		nodes[etid * 3][ltid] = temp[0];
		nodes[etid * 3 + 1][ltid] = temp[1];
		nodes[etid * 3 + 2][ltid] = temp[2];

		//START OF SECOND BATCH//////////////////////////
		index = t_ele->nodeindex[etid+THREADS_PER_ELE][ltid];

		temp2[0] = x[index];
		temp2[1] = x[index + numnodes];
		temp2[2] = x[index + numnodes * 2];

		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			temp[j] = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
			temp[j] += R[k][j][ltid] * temp2[k];
		}

		nodes[(etid+THREADS_PER_ELE) * 3][ltid] = temp[0];
		nodes[(etid+THREADS_PER_ELE) * 3 + 1][ltid] = temp[1];
		nodes[(etid+THREADS_PER_ELE) * 3 + 2][ltid] = temp[2];

		////////////////////////////////////////////////////////////
		
		if(etid < 2)
		{
			index = t_ele->nodeindex[etid+THREADS_PER_ELE * 2][ltid];

			temp2[0] = x[index];
			temp2[1] = x[index + numnodes];
			temp2[2] = x[index + numnodes * 2];

			#pragma unroll 3
			for(int j=0;j<3;j++)
			{
				temp[j] = 0;
				#pragma unroll 3
				for(int k=0;k<3;k++)
				temp[j] += R[k][j][ltid] * temp2[k];
			}

			nodes[(etid+THREADS_PER_ELE * 2) * 3][ltid] = temp[0];
			nodes[(etid+THREADS_PER_ELE * 2) * 3 + 1][ltid] = temp[1];
			nodes[(etid+THREADS_PER_ELE * 2) * 3 + 2][ltid] = temp[2];
		}		
	}

	__syncthreads();

	if(tid < numelements)
	{
		#pragma unroll 30
		for(int i=0;i<30;i++)
			out[i] = nodes[i][ltid];

		mulK(out, B, t_ele->c1, t_ele->c2);

		if(etid == 0)
			#pragma unroll 30
			for(int i=0;i<30;i++)
				nodes[i][ltid] = out[i];
	}

	__syncthreads();

	if(tid < numelements && etid == 1)
	{
		#pragma unroll 30
		for(int i=0;i<30;i++)
			nodes[i][ltid] += out[i];
	}

	__syncthreads();

	if(tid < numelements && etid == 2)
	{
		#pragma unroll 30
		for(int i=0;i<30;i++)
			nodes[i][ltid] += out[i];
	}

	__syncthreads();

	if(tid < numelements && etid == 3)
	{
			#pragma unroll 30
			for(int i=0;i<30;i++)
				nodes[i][ltid] += out[i];
	}

	__syncthreads();

	if(tid < numelements)
	{
		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			float temp3 = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
				temp3 += R[j][k][ltid] * nodes[(etid)*3 + k][ltid];

			t_solvedata->product[(etid)*3 + j][ltid] = temp3;
		}

		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			float temp3 = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
				temp3 += R[j][k][ltid] * nodes[(etid+THREADS_PER_ELE)*3 + k][ltid];

			t_solvedata->product[(etid+THREADS_PER_ELE)*3 + j][ltid] = temp3;
		}

		if(etid <2)
		{
			#pragma unroll 3
			for(int j=0;j<3;j++)
			{
				float temp3 = 0;
				#pragma unroll 3
				for(int k=0;k<3;k++)
					temp3 += R[j][k][ltid] * nodes[(etid+THREADS_PER_ELE*2)*3 + k][ltid];

				t_solvedata->product[(etid+THREADS_PER_ELE*2)*3 + j][ltid] = temp3;
			}
		}


	}

}


#endif






__device__
void dot(float*a, float*b, float* out, int n) 
{
	__shared__ float temp[DOT_BLOCK_SIZE];
	int index = threadIdx.x;
	int element = index;

	float tmp = 0;

	while(element < n)
	{
		tmp += a[element] * b[element];
		element += DOT_BLOCK_SIZE;
	}

	temp[index] = tmp;

	__syncthreads();


	int i = DOT_BLOCK_SIZE >> 1;
	while(i>0)
	{
		if(index < i)
			temp[index] += temp[index + i];
		__syncthreads();
		i>>=1;
	}

	if(index == 0)
		*out = temp[0];
}



//step 1
//precompute
__global__
void precompute(GPUElement* elements, mulData* solverData, float* xt, int numelements, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	int bid = blockIdx.x;
	int ltid = threadIdx.x;

	if(tid < numelements)
	{
		GPUElement* t_ele = &(elements[bid]);
		mulData* t_solvedata = &(solverData[bid]);

		float nodes[4], R[3][3]={0}, D[3][3];
			

		#pragma unroll 4
		for(int i=0;i<4;i++)
		{
			int index = t_ele->nodeindex[i][ltid];
			nodes[i * 3] = xt[index];
			nodes[i * 3 + 1] = xt[index + numnodes];
			nodes[i * 3 + 2] = xt[index + numnodes * 2];
		}

		#pragma unroll 3
		for(int i=0;i<3;i++)
			#pragma unroll 3
			for(int j=0;j<3;j++)
				D[i][j] = t_ele->B[i][j][ltid];

		#pragma unroll 3
		for(int i=0;i<3;i++)
			#pragma unroll 3
			for(int j=0;j<3;j++)
					R[i][j] = (nodes[i] - nodes[9 + i]) * D[0][j] + 
							  (nodes[3 + i] - nodes[9 + i]) * D[1][j] + 
							  (nodes[6 + i] - nodes[9 + i]) * D[2][j];

		gpuComputePolarDecomposition(R);
	
		#pragma unroll 3
		for(int i=0;i<3;i++)
			#pragma unroll 3
			for(int j=0;j<3;j++)
				t_solvedata->R[i][j][ltid] =  R[i][j];

		float temp[3];
		float temp2[3];

		#pragma unroll 10
		for(int i=0;i<10;i++)
		{
			temp[0] = t_ele->f0[i * 3][ltid];
			temp[1] = t_ele->f0[i * 3 + 1][ltid];
			temp[2] = t_ele->f0[i * 3 + 2][ltid];

			#pragma unroll 3
			for(int j=0;j<3;j++)
			{
				temp2[j] = 0;
				#pragma unroll 3
				for(int k=0;k<3;k++)
					temp2[j] += R[j][k] * temp[k];
			}
			
			t_solvedata->b[i * 3][ltid] = temp2[0];
			t_solvedata->b[i * 3 + 1][ltid] = temp2[1];	
			t_solvedata->b[i * 3 + 2][ltid] = temp2[2];	

		}
			
		
	}
}

//step 1.5
//precompute
__global__
void
makeRKRT(GPUElement* elements, mulData* solverData, float* x, int numelements, int numnodes)
{
		mulSystem(elements, solverData, x, numelements, numnodes);
}

//step 2
//precompute
__global__
void gatherB(GPUNode* nodes, mulData* solverData, float* b, float* mass, float* vt, float* extforces, char* allowed,int numnodes)
{
	int groupid = threadIdx.x % NODE_BLOCK_SIZE;// / NODE_THREADS;
	int grouptid = threadIdx.x / NODE_BLOCK_SIZE; //% NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_THREADS][NODE_BLOCK_SIZE][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[grouptid][groupid];
	
	if(nodeno < numnodes)
	{

		cache[grouptid][groupid][0] = 0;
		cache[grouptid][groupid][1] = 0;
		cache[grouptid][groupid][2] = 0;


		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][grouptid][groupid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][grouptid][groupid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][grouptid][groupid];

			cache[grouptid][groupid][0] += solverData[tetindex].b[nodeindex * 3][tetindex2] - solverData[tetindex].product[nodeindex * 3][tetindex2];
			cache[grouptid][groupid][1] += solverData[tetindex].b[nodeindex * 3 + 1][tetindex2] - solverData[tetindex].product[nodeindex * 3 + 1][tetindex2];
			cache[grouptid][groupid][2] += solverData[tetindex].b[nodeindex * 3 + 2][tetindex2] - solverData[tetindex].product[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{
			b[nodeno]     = (cache[0][groupid][0] + cache[1][groupid][0]) * dt + mass[nodeno] * vt[nodeno] + extforces[nodeno] * dt;
			b[nodeno + numnodes] = (cache[0][groupid][1] + cache[1][groupid][1]) * dt + mass[nodeno + numnodes] * vt[nodeno + numnodes] + extforces[nodeno + numnodes] * dt;
			b[nodeno + numnodes * 2] = (cache[0][groupid][2] + cache[1][groupid][2]) * dt + mass[nodeno + numnodes * 2] * vt[nodeno + numnodes * 2] + extforces[nodeno + numnodes * 2] * dt;

			char bitsy = allowed[nodeno];
			if(bitsy & 1)
				vt[nodeno] = 0;
			if(bitsy & 2)
				vt[nodeno + numnodes] = 0;
			if(bitsy & 4)
				vt[nodeno + numnodes * 2] = 0;

		}
	}
}


#ifdef _GAUSSIAN_QUADRATURE_

__global__
void
makeMinv(GPUElement* elements, mulData* solverData, int numelements)
{
	int bid = blockIdx.x;
	int ltid = threadIdx.x % BLOCK_SIZE;
	int etid = threadIdx.x / BLOCK_SIZE;
	int tid = ltid + blockIdx.x * BLOCK_SIZE;

	__shared__ float R[3][3][BLOCK_SIZE];
	__shared__ float b[4][3][BLOCK_SIZE];
	__shared__ float c1[BLOCK_SIZE];
	__shared__ float c2[BLOCK_SIZE];
	__shared__ float final[30][BLOCK_SIZE];
	
	float minv[30];

	GPUElement* t_ele = &(elements[bid]);
	mulData* t_solvedata = &(solverData[bid]);

	if(tid < numelements)
	{
		if(etid == 0)
		{
			#pragma unroll 3
			for(int i=0;i<3;i++)
				#pragma unroll 3
				for(int j=0;j<3;j++)
					R[i][j][ltid] = t_solvedata->R[i][j][ltid];

			#pragma unroll 3
			for(int i=0;i<3;i++)
				#pragma unroll 3
				for(int j=0;j<3;j++)
					b[i][j][ltid] = t_ele->B[i][j][ltid];

			b[3][0][ltid] = -b[0][0][ltid]-b[1][0][ltid]-b[2][0][ltid];
			b[3][1][ltid] = -b[0][1][ltid]-b[1][1][ltid]-b[2][1][ltid];
			b[3][2][ltid] = -b[0][2][ltid]-b[1][2][ltid]-b[2][2][ltid];

			c1[ltid] = t_ele->c1[ltid];
			c2[ltid] = t_ele->c2[ltid];
		}
	}
	__syncthreads();

	if(tid < numelements)
	{
		
		float con1 = c1[ltid];
		float con2 = c2[ltid];
		float con3 = (con1 - con2)/2.0;
		float* s = &S[etid * 4];

		float dndx[3][10];
					
		dndx[0][0] = (s[0]-1.0) * b[0][0][ltid];
		dndx[0][1] = (s[1]-1.0) * b[1][0][ltid];
		dndx[0][2] = (s[2]-1.0) * b[2][0][ltid];
		dndx[0][3] = (s[3]-1.0) * b[3][0][ltid];
		dndx[0][4] = (b[1][0][ltid] * s[0] +  b[0][0][ltid] * s[1]);
		dndx[0][5] = (b[2][0][ltid] * s[1] +  b[1][0][ltid] * s[2]);
		dndx[0][6] = (b[0][0][ltid] * s[2] +  b[2][0][ltid] * s[0]);
		dndx[0][7] = (b[3][0][ltid] * s[0] +  b[0][0][ltid] * s[3]);
		dndx[0][8] = (b[3][0][ltid] * s[1] +  b[1][0][ltid] * s[3]);
		dndx[0][9] = (b[3][0][ltid] * s[2] +  b[2][0][ltid] * s[3]);
	
		dndx[1][0] = (s[0]-1.0) * b[0][1][ltid];
		dndx[1][1] = (s[1]-1.0) * b[1][1][ltid];
		dndx[1][2] = (s[2]-1.0) * b[2][1][ltid];
		dndx[1][3] = (s[3]-1.0) * b[3][1][ltid];
		dndx[1][4] = (b[1][1][ltid] * s[0] +  b[0][1][ltid] * s[1]);
		dndx[1][5] = (b[2][1][ltid] * s[1] +  b[1][1][ltid] * s[2]);
		dndx[1][6] = (b[0][1][ltid] * s[2] +  b[2][1][ltid] * s[0]);
		dndx[1][7] = (b[3][1][ltid] * s[0] +  b[0][1][ltid] * s[3]);
		dndx[1][8] = (b[3][1][ltid] * s[1] +  b[1][1][ltid] * s[3]);
		dndx[1][9] = (b[3][1][ltid] * s[2] +  b[2][1][ltid] * s[3]);
	
		dndx[2][0] = (s[0]-1.0) * b[0][2][ltid];
		dndx[2][1] = (s[1]-1.0) * b[1][2][ltid];
		dndx[2][2] = (s[2]-1.0) * b[2][2][ltid];
		dndx[2][3] = (s[3]-1.0) * b[3][2][ltid];
		dndx[2][4] = (b[1][2][ltid] * s[0] +  b[0][2][ltid] * s[1]);
		dndx[2][5] = (b[2][2][ltid] * s[1] +  b[1][2][ltid] * s[2]);
		dndx[2][6] = (b[0][2][ltid] * s[2] +  b[2][2][ltid] * s[0]);
		dndx[2][7] = (b[3][2][ltid] * s[0] +  b[0][2][ltid] * s[3]);
		dndx[2][8] = (b[3][2][ltid] * s[1] +  b[1][2][ltid] * s[3]);
		dndx[2][9] = (b[3][2][ltid] * s[2] +  b[2][2][ltid] * s[3]);

		float mat[3][3];
		float diag[3][3];
		float temp;

		#pragma unroll 10
		for(int i=0;i<10;i++)
		{
			diag[0][0] = dndx[0][i] * dndx[0][i] * con1 + dndx[1][i] * dndx[1][i] * con3 + dndx[2][i] * dndx[2][i] * con3;
			diag[1][1] = dndx[1][i] * dndx[1][i] * con1 + dndx[0][i] * dndx[0][i] * con3 + dndx[2][i] * dndx[2][i] * con3;
			diag[2][2] = dndx[2][i] * dndx[2][i] * con1 + dndx[1][i] * dndx[1][i] * con3 + dndx[0][i] * dndx[0][i] * con3;

			diag[0][1] = dndx[0][i] * dndx[1][i] * con2 + dndx[0][i] * dndx[1][i] * con3;
			diag[0][2] = dndx[0][i] * dndx[2][i] * con2 + dndx[0][i] * dndx[2][i] * con3;
			diag[1][2] = dndx[1][i] * dndx[2][i] * con2 + dndx[1][i] * dndx[2][i] * con3;

			diag[1][0] = diag[0][1];
			diag[2][0] = diag[0][2];
			diag[2][1] = diag[1][2];

			#pragma unroll 3
			for(int j=0;j<3;j++)
				#pragma unroll 3
				for(int k=0;k<3;k++)
				{
					mat[j][k] = 0;
					#pragma unroll 3
					for(int l=0;l<3;l++)
						mat[j][k] += diag[j][l] * R[k][l][ltid]; //RT
				}
			
			#pragma unroll 3
			for(int j=0;j<3;j++)
			{
				temp = 0;
				#pragma unroll 3
				for(int k=0;k<3;k++)
					temp +=R[j][k][ltid] * mat[k][j];
				minv[i * 3 + j] = temp;
			}
		}

		if(etid == 0)
			#pragma unroll 30
			for(int i=0;i<30;i++)
				final[i][ltid] = minv[i];
	}

	__syncthreads();

	if(tid < numelements && etid == 1)
	{
		#pragma unroll 30
		for(int i=0;i<30;i++)
			final[i][ltid] += minv[i];
	}

	__syncthreads();

	if(tid < numelements && etid == 2)
	{
		#pragma unroll 30
		for(int i=0;i<30;i++)
			final[i][ltid] += minv[i];
	}

	__syncthreads();

	if(tid < numelements && etid == 3)
	{
		#pragma unroll 30
		for(int i=0;i<30;i++)
			final[i][ltid] += minv[i];
	}

	__syncthreads();

	if(tid < numelements)
	{
		#pragma unroll 3
		for(int j=0;j<3;j++)
			t_solvedata->product[etid*3 + j][ltid] = final[etid*3 + j][ltid];

		#pragma unroll 3
		for(int j=0;j<3;j++)
			t_solvedata->product[(etid+THREADS_PER_ELE)*3 + j][ltid] = final[(etid+THREADS_PER_ELE)*3 + j][ltid];

		if(etid <2)
		{
			#pragma unroll 3
			for(int j=0;j<3;j++)
				t_solvedata->product[(etid+THREADS_PER_ELE * 2)*3 + j][ltid] = final[(etid+THREADS_PER_ELE * 2)*3 + j][ltid];
		}
	}

}

__global__
void gatherMinv(GPUNode* nodes, mulData* solverData, float* mass, float* minv, int numnodes)
{
	int groupid = threadIdx.x % NODE_BLOCK_SIZE;// / NODE_THREADS;
	int grouptid = threadIdx.x / NODE_BLOCK_SIZE; //% NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_THREADS][NODE_BLOCK_SIZE][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[grouptid][groupid];
	
	if(nodeno < numnodes)
	{

		cache[grouptid][groupid][0] = 0;
		cache[grouptid][groupid][1] = 0;
		cache[grouptid][groupid][2] = 0;


		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][grouptid][groupid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][grouptid][groupid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][grouptid][groupid];

			cache[grouptid][groupid][0] += solverData[tetindex].product[nodeindex * 3][tetindex2];
			cache[grouptid][groupid][1] += solverData[tetindex].product[nodeindex * 3 + 1][tetindex2];
			cache[grouptid][groupid][2] += solverData[tetindex].product[nodeindex * 3 + 2][tetindex2];
		}
}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{
			float m1 = (cache[0][groupid][0] + cache[1][groupid][0]) * COEFFK + mass[nodeno] * COEFFM;
			float m2 = (cache[0][groupid][1] + cache[1][groupid][1]) * COEFFK + mass[nodeno + numnodes] * COEFFM;
			float m3 = (cache[0][groupid][2] + cache[1][groupid][2]) * COEFFK + mass[nodeno + numnodes * 2] * COEFFM;


			minv[nodeno]     = 1.0/(fabs(m1) > ZERO_EPS ? m1 : 1.0);
			minv[nodeno + numnodes] = 1.0/(fabs(m2) > ZERO_EPS ? m2 : 1.0);
			minv[nodeno + numnodes * 2] = 1.0/(fabs(m3) > ZERO_EPS ? m3 : 1.0);
		}
	}
}

#endif

//step 1
//init CG
// x = velocity
__global__
void
initAx(GPUElement* elements, mulData* solverData, float* x, int numelements, int numnodes)
{
		mulSystem(elements, solverData, x, numelements, numnodes);
}

//step2
//init CG
__global__
void
initRandD(GPUNode* nodes, mulData* solverData, float* r, float* d, float* b, float* mass, float* vt, char* allowed, 
			float* minv,
			int numnodes)
{
	int groupid = threadIdx.x % NODE_BLOCK_SIZE;// / NODE_THREADS;
	int grouptid = threadIdx.x / NODE_BLOCK_SIZE; //% NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_THREADS][NODE_BLOCK_SIZE][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[grouptid][groupid];
	
	if(nodeno < numnodes)
	{

		cache[grouptid][groupid][0] = 0;
		cache[grouptid][groupid][1] = 0;
		cache[grouptid][groupid][2] = 0;

		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][grouptid][groupid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][grouptid][groupid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][grouptid][groupid];

			cache[grouptid][groupid][0] += solverData[tetindex].product[nodeindex * 3][tetindex2];
			cache[grouptid][groupid][1] += solverData[tetindex].product[nodeindex * 3 + 1][tetindex2];
			cache[grouptid][groupid][2] += solverData[tetindex].product[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{	
			char bitsy = allowed[nodeno];

			//r = b-Ax

	#if defined(_GAUSSIAN_QUADRATURE_)
			float r0 =  (bitsy & 1) ? 0 : (b[nodeno] - ( (cache[0][groupid][0] + cache[1][groupid][0]) * COEFFK + mass[nodeno] * vt[nodeno] * COEFFM)) * minv[nodeno];
			float r1 =  (bitsy & 2) ? 0 : (b[nodeno + numnodes] - ( (cache[0][groupid][1] + cache[1][groupid][1]) * COEFFK + mass[nodeno + numnodes] * vt[nodeno + numnodes] * COEFFM)) * minv[nodeno + numnodes];
			float r2 =  (bitsy & 4) ? 0 : (b[nodeno + numnodes * 2] - ( (cache[0][groupid][2] + cache[1][groupid][2]) * COEFFK + mass[nodeno + numnodes * 2] * vt[nodeno + numnodes * 2] * COEFFM)) * minv[nodeno + numnodes * 2];
	#else
			float r0 =  (bitsy & 1) ? 0 : (b[nodeno] - ( (cache[0][groupid][0] + cache[1][groupid][0]) * COEFFK + mass[nodeno] * vt[nodeno] * COEFFM));
			float r1 =  (bitsy & 2) ? 0 : (b[nodeno + numnodes] - ( (cache[0][groupid][1] + cache[1][groupid][1]) * COEFFK + mass[nodeno + numnodes] * vt[nodeno + numnodes] * COEFFM));
			float r2 =  (bitsy & 4) ? 0 : (b[nodeno + numnodes * 2] - ( (cache[0][groupid][2] + cache[1][groupid][2]) * COEFFK + mass[nodeno + numnodes * 2] * vt[nodeno + numnodes * 2] * COEFFM));
	#endif
			r[nodeno] = r0;
			r[nodeno + numnodes] = r1;
			r[nodeno + numnodes * 2] = r2;

			//d=r
			d[nodeno] = r0;
			d[nodeno + numnodes] = r1;
			d[nodeno + numnodes * 2] = r2;
		}
	}

}

//step3
//init CG
//1 block, BLOCK_SIZE threads
__global__
void
initDeltaVars(CGVars* vars, float* r, int numnodes)
{
	__shared__ float rr;
	dot(r, r, &rr, numnodes * 3);
	
	if(threadIdx.x == 0)
	{
		vars->deltaNew = rr;
		vars->delta0 = vars->deltaNew;
	}
}

//step 4
//CG loop
//q = Ad
__global__
void
makeQprod(GPUElement* elements, mulData* solverData, float* d, int numelements, int numnodes)
{
		mulSystem(elements, solverData, d, numelements, numnodes);
}

//step 5
//CG loop
//q = Ad
__global__
void
gatherQprod(GPUNode* nodes, mulData* solverData, float* q, float* mass, float* d, char* allowed, 
			float* minv, 
			int numnodes)
{
	int groupid = threadIdx.x % NODE_BLOCK_SIZE;// / NODE_THREADS;
	int grouptid = threadIdx.x / NODE_BLOCK_SIZE; //% NODE_THREADS;
	int nodeno = blockIdx.x * NODE_BLOCK_SIZE + groupid;

	__shared__ float cache[NODE_THREADS][NODE_BLOCK_SIZE][3];
	GPUNode* node = &(nodes[blockIdx.x]);
	int n = node->n[grouptid][groupid];
	
	if(nodeno < numnodes)
	{

		cache[grouptid][groupid][0] = 0;
		cache[grouptid][groupid][1] = 0;
		cache[grouptid][groupid][2] = 0;

		for(int i=0;i<n;i++)
		{
			int tetindex = node->elementindex[i][0][grouptid][groupid] / BLOCK_SIZE;
			int tetindex2 = node->elementindex[i][0][grouptid][groupid] % BLOCK_SIZE;
			int nodeindex = node->elementindex[i][1][grouptid][groupid];

			cache[grouptid][groupid][0] += solverData[tetindex].product[nodeindex * 3][tetindex2];
			cache[grouptid][groupid][1] += solverData[tetindex].product[nodeindex * 3 + 1][tetindex2];
			cache[grouptid][groupid][2] += solverData[tetindex].product[nodeindex * 3 + 2][tetindex2];
		}
	}

	__syncthreads();

	if(nodeno < numnodes)
	{
		if(grouptid == 0)
		{
			char bitsy = allowed[nodeno];
	#if defined(_GAUSSIAN_QUADRATURE_)
			q[nodeno]     = (bitsy & 1) ? 0 : ( (cache[0][groupid][0] + cache[1][groupid][0]) * COEFFK + mass[nodeno] * d[nodeno] * COEFFM) * minv[nodeno];
			q[nodeno + numnodes] = (bitsy & 2) ? 0 : ( (cache[0][groupid][1] + cache[1][groupid][1]) * COEFFK + mass[nodeno + numnodes] * d[nodeno + numnodes] * COEFFM) * minv[nodeno + numnodes];
			q[nodeno + numnodes * 2] = (bitsy & 4) ? 0 : ( (cache[0][groupid][2] + cache[1][groupid][2])  * COEFFK + mass[nodeno + numnodes * 2] * d[nodeno + numnodes * 2] * COEFFM) * minv[nodeno + numnodes*2];
	#else
			q[nodeno]     = (bitsy & 1) ? 0 : ( (cache[0][groupid][0] + cache[1][groupid][0]) * COEFFK + mass[nodeno] * d[nodeno] * COEFFM);
			q[nodeno + numnodes] = (bitsy & 2) ? 0 : ( (cache[0][groupid][1] + cache[1][groupid][1]) * COEFFK + mass[nodeno + numnodes] * d[nodeno + numnodes] * COEFFM);
			q[nodeno + numnodes * 2] = (bitsy & 4) ? 0 : ( (cache[0][groupid][2] + cache[1][groupid][2])  * COEFFK + mass[nodeno + numnodes * 2] * d[nodeno + numnodes * 2] * COEFFM);
	#endif
		}
	}

}

//step 6
//CG Loop
//make vars
//1 block, BLOCK_SIZE threads
__global__
void
makeVars(CGVars* vars, float* d, float* q, float* r, int numnodes)
{
	float dq, rq, qq;
	dot(d,q,&dq,numnodes * 3);
	dot(r,q,&rq,numnodes * 3);
	dot(q,q,&qq,numnodes * 3);

	__syncthreads();

	if(threadIdx.x == 0)
	{
		vars->alpha = vars->deltaNew / dq;
		vars->deltaOld = vars->deltaNew;

		//r.r = r'.r' - 2*alpha*(r'.q) + alpha * alpha * (q.q)
		vars->deltaNew = vars->deltaNew - (2 * vars->alpha) * rq + (vars->alpha * vars->alpha) * qq;
		vars->beta = vars->deltaNew / vars->deltaOld;
	}
}

//step 7
//CG Loop
//make x, r, d
//x = velocity
__global__
void
makeXRandD(CGVars* vars, float *x, float* r, float* d, float* q, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x * VECTOR_BLOCK_SIZE;
	if(tid < numnodes)
	{
		float alpha = vars->alpha;
		float beta = vars->beta;
		float d1,d2,d3;
		float r1,r2,r3;

		d1 = d[tid];
		d2 =  d[tid + numnodes];
		d3 = d[tid + numnodes * 2];

		x[tid] = x[tid] + alpha * d1;
		x[tid + numnodes] = x[tid + numnodes] + alpha * d2;
		x[tid + numnodes * 2] = x[tid + numnodes * 2] + alpha * d3;

		r1 = r[tid] - alpha * q[tid];
		r2 = r[tid + numnodes] - alpha * q[tid + numnodes];
		r3 = r[tid + numnodes * 2] - alpha * q[tid + numnodes * 2];

		d[tid] = r1 + beta * d1;
		d[tid + numnodes] = r2 + beta * d2;
		d[tid + numnodes * 2] = r3 + beta * d3;

		r[tid] = r1;
		r[tid + numnodes] = r2;
		r[tid + numnodes * 2] = r3;
	}
} 

//step 8
//make x(t+1)
__global__
void
integrate(float *x, float* v, int numnodes)
{
	int tid = threadIdx.x + blockIdx.x * VECTOR_BLOCK_SIZE;
	if(tid < numnodes)
	{
		x[tid] = x[tid] + dt * v[tid];
		x[tid + numnodes] = x[tid + numnodes] + dt * v[tid + numnodes];
		x[tid + numnodes * 2] = x[tid + numnodes * 2] + dt * v[tid + numnodes * 2];
	}
}

__host__
void
gpuTimeStep(int numelements, int numnodes)
{
	const int num_blocks_ele = (numelements/BLOCK_SIZE) + 1;
	const int num_blocks_node = (numnodes/NODE_BLOCK_SIZE) + 1;
	const int num_blocks_vec = (numnodes/VECTOR_BLOCK_SIZE) + 1;

	hipError_t error;

	printf("Started\n");
	
	precompute<<<num_blocks_ele, BLOCK_SIZE>>>(gpuptrElements, gpuptrMulData, gpuptr_xt, numelements, numnodes);
	
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("1");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	makeRKRT<<<num_blocks_ele, THREADS_PER_BLOCK>>>(gpuptrElements, gpuptrMulData, gpuptr_xt, numelements, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("1.5");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	gatherB<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptr_b, gpuptr_mass, gpuptr_vt, gpuptr_extforces, gpuptr_allowed, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("2");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	//inspectGPUBuffer(gpuptr_b,numnodes);

	
	#ifdef _GAUSSIAN_QUADRATURE_
	makeMinv<<<num_blocks_ele, THREADS_PER_BLOCK>>>(gpuptrElements, gpuptrMulData, numelements);
	gatherMinv<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptr_mass, gpuptr_minv, numnodes);
	#endif

	initAx<<<num_blocks_ele, THREADS_PER_BLOCK>>>(gpuptrElements, gpuptrMulData, gpuptr_vt, numelements, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("3");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	initRandD<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptrR, gpuptrD, gpuptr_b, gpuptr_mass, gpuptr_vt,  gpuptr_allowed,
		gpuptr_minv,
		numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("4");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	initDeltaVars<<<1, DOT_BLOCK_SIZE>>>(gpuptrVars, gpuptrR, numnodes);

	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		printf("5");
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		//exit(-1);
	}

	int i=0;

	CGVars vars;
	hipMemcpy(&vars, gpuptrVars, sizeof(CGVars), hipMemcpyDeviceToHost);

	printf("Loop Started");
	
	while(i < MAX_ITER && vars.deltaNew > (EPSIL * EPSIL) * vars.delta0)
	{
		makeQprod<<<num_blocks_ele, THREADS_PER_BLOCK>>>(gpuptrElements, gpuptrMulData, gpuptrD, numelements, numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("6");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		gatherQprod<<<num_blocks_node, GATHER_THREAD_NO>>>(gpuptrNodes, gpuptrMulData, gpuptrQ, gpuptr_mass, gpuptrD, gpuptr_allowed,
			gpuptr_minv,
			numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("7");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		makeVars<<<1, DOT_BLOCK_SIZE>>>(gpuptrVars, gpuptrD, gpuptrQ, gpuptrR, numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("8");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		makeXRandD<<<num_blocks_vec, VECTOR_BLOCK_SIZE>>>(gpuptrVars, gpuptr_vt, gpuptrR, gpuptrD, gpuptrQ, numnodes);

		hipDeviceSynchronize();
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			printf("9");
			// print the CUDA error message and exit
			printf("CUDA error: %s\n", hipGetErrorString(error));
			//exit(-1);
		}

		hipMemcpy(&vars, gpuptrVars, sizeof(CGVars), hipMemcpyDeviceToHost);
		i++;

	}
	
	printf("Loop Ended: %d\n", i);

	integrate<<<num_blocks_vec, VECTOR_BLOCK_SIZE>>>(gpuptr_xt, gpuptr_vt, numnodes);
}


#endif


/* EXTRAS


__device__
void makeFU(float f0[30][BLOCK_SIZE], float R[3][3], float out[30])
{
	int ltid = threadIdx.x;
	float x[30];

	#pragma unroll 30
	for(int i=0;i<30;i++)
		x[i] = f0[i][ltid];

	#pragma unroll 10
	for(int i=0;i<10;i++)
		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			out[i*3 + j] = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
			out[i*3+j] += R[j][k] * x[i*3 + k];
		}		
}

//xt will be malnipulated and used as a temp array. do not further malnipulate xt
__device__
void makeRKRT(float system[30][30][BLOCK_SIZE], float R[3][3], float xt[30], float b[30])
{
	float temp[30];

	#pragma unroll 10
	for(int i=0;i<10;i++)
		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			temp[i*3 + j] = 0;
			#pragma unroll 3
			for(int k=0;k<3;k++)
			temp[i*3+j] += R[k][j] * xt[i*3 + k]; //RT first
		}

	#pragma unroll 30
	for(int i=0;i<30;i++)
		xt[i] = 0;
	
	#pragma unroll 30
	for(int j=0;j<30;j++)
	{
		#pragma unroll 30
		for(int i=0;i<30;i++)
		xt[i] += system[j][i][threadIdx.x] * temp[j];
	}		

	#pragma unroll 10
	for(int i=0;i<10;i++)
		#pragma unroll 3
		for(int j=0;j<3;j++)
		{
			#pragma unroll 3
			for(int k=0;k<3;k++)
			b[i*3+j] -= R[j][k] * xt[i*3 + k];
		}

}

*/